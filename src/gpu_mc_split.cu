#include "hip/hip_runtime.h"
//
// Created by davide on 5/3/24.
//

#include "test.hpp"
#include <vector>
#include <string.h>
#include <string>
#include <queue>
#include <algorithm>
#include <unordered_set>

using namespace std;
const int DIM_POOL = 16;
std::vector<std::vector<float>> g0;
std::vector<std::vector<float>> g1;
std::vector<float> edge_labels;
vector<pair<int,int>> m_best;

__device__ float *gpu_edge_labels;
__device__ int size_edge_labels;

__device__ float **gpu_g0;
__device__ int size_gpu_g0_row;
__device__ int size_gpu_g0_col;

__device__ float **gpu_g1;
__device__ int size_gpu_g1_row;
__device__ int size_gpu_g1_col;

typedef struct{
    int g_size;
    int h_size;
    int row_ring_size;
    int *col_ring_size;
    int *g;
    int *h;
    int adj;
    char label[4];
    int **rings_g;
}GpuLabelClass;


typedef struct{
    int first;
    int second;
}Pair;


typedef struct {
    int labels_size;
    int m_size;
    GpuLabelClass *labels;
    Pair *m_local;
}ThreadVar;

// vtx_set: selected label class
// g: selected graph
int select_vertex(std::vector<int>& vtx_set, std::vector<std::vector<float> >& g) {
    // selects node from graph given a label, choosing an adjacent node with the maximum degree

    int max_deg = -1;
    int vtx = 0;
    for (int c_vtx : vtx_set) {
        int deg = 0;
        for (float i : g[c_vtx]) {
            if (i != 0) {
                deg += 1;
            }
        }

        if (deg > max_deg) {
            max_deg = deg;
            vtx = c_vtx;
        }
    }
    return vtx;
}
std::vector<std::string> find_common_strings(const std::vector<std::string>& l0, const std::vector<std::string>& l1) {
    // Utilizzare un set per memorizzare ed effettuare velocemente la ricerca di stringhe comuni uniche
    std::unordered_set<std::string> common_strings(l0.begin(), l0.end());

    // Vettore per memorizzare le stringhe comuni trovate
    std::vector<std::string> result;

    // Trovare le intersezioni tra le stringhe della seconda lista e le stringhe nel set
    for (const std::string& str : l1) {
        // Se la stringa è presente nel set delle stringhe comuni
        if (common_strings.find(str) != common_strings.end()) {
            // Aggiungila al risultato
            result.push_back(str);
            // Rimuovi la stringa dal set per evitare duplicati
            common_strings.erase(str);
        }
    }
    return result;
}
std::vector<LabelClass> cpu_gen_initial_labels(const std::vector<std::string>& l0, const std::vector<std::string>& l1,     std::vector<std::vector<int> >& ring_classes){
    std::vector<LabelClass> label_classes;
    const std::vector<string> common_labels = find_common_strings(l0,l1);


    for (const std::string& label : common_labels) {
        // Filter atoms and ring data based on label
        std::vector<int> g_elems;
        std::vector<std::vector<int> > g_ring_classes;
        for (size_t i = 0; i < l0.size(); ++i) {
            if (l0[i] == label) {
                g_elems.push_back(i);
                if( !ring_classes.empty() ) g_ring_classes.push_back(ring_classes[i]); // Assuming ring_classes access by index

            }
        }

        std::vector<int> h_elems;
        for (size_t j = 0; j < l1.size(); ++j) {
            if (l1[j] == label) {
                h_elems.push_back(j);
            }
        }

        LabelClass label_tmp(g_elems,h_elems,g_ring_classes,0, label);
        label_classes.push_back(label_tmp);

    }
    return label_classes;
}
std::vector<float> cpu_gen_bond_labels(const std::vector<std::vector<float> >& g0, const std::vector<std::vector<float> >& g1) {

    std::vector<float> all_labels;
    all_labels.push_back(0.0);
    all_labels.push_back(1.0);
    all_labels.push_back(1.5);
    all_labels.push_back(2.0);
    all_labels.push_back(3.0);
    all_labels.push_back(4.0);
    all_labels.push_back(5.0);
    all_labels.push_back(6.0);
    // Vector to store potential bond labels (extracted from both matrices)
    std::vector<float> intersection;

    // Iterate over rows and columns of g0
    for (size_t i = 0; i < g0.size(); ++i) {
        for (size_t j = 0; j < g0[i].size(); ++j) {
            float current_label = g0[i][j];

            // Check if the label exists in each row of g1 (avoid nested loops)
            bool found_in_g1 = false;
            for (const std::vector<float>& row : g1) {
                if (std::find(row.begin(), row.end(), current_label) != row.end()) {
                    found_in_g1 = true;
                    break;
                }
            }

            // If found in g1 and not already in the intersection, add it
            if (found_in_g1 && (intersection.empty() || intersection.back() != current_label)) {
                intersection.push_back(current_label);
            }
        }
    }


    // Sort the intersection vector for desired output
    std::sort(intersection.begin(), intersection.end());
    // Use unique to remove consecutive duplicates (may leave gaps)
    intersection.erase(std::unique(intersection.begin(), intersection.end()), intersection.end());

    // Resize the vector to remove empty space from erasing (optional)
    intersection.resize(intersection.size());

    // Return the intersection vector containing common bond labels (sorted and unique)
    return intersection;
}
std::vector<int> hood(int vtx, const std::vector<std::vector<float>>& g, float edge) {
    // Return the neighbors of a specified node, with the specified bond type.
    std::vector<int> friends;
    for (std::size_t i = 0; i < g.size(); ++i) {
        if (g[i][vtx] == edge && static_cast<std::size_t>(vtx) != i) {
            friends.push_back(i);
        }
    }

    return friends;
}




__device__
bool contains(int value, int *arr, int size) {
    for (int i = 0; i < size; ++i) {
        if (arr[i] == value) {
            return true;
        }
    }
    return false;
}


bool host_contains(int value, int *arr, int size) {
    for (int i = 0; i < size; ++i) {
        if (arr[i] == value) {
            return true;
        }
    }
    return false;
}

//      puts into a 2D array the data regarding indexes of rings related to the array of elements
//      2D-array that will contain the result that will be modified
//      1D-array containing idxList
//      1D-array of elements
//      int size of elements
__device__
int kernel_get_ring_match_data(int *dim_col, int **result, int *idxList, int *elems, int elem_size, GpuLabelClass *lc){
    int index;

    for( int i = 0; i < elem_size ; ++i){
        for( int j = 0 ; j < lc->g_size ; ++j ){
            if( lc->g[j] == elems[i] ) idxList[index] = index;
            index++;
        }
    }
    for ( int k = 0 , i = 0 ; k < index+1 ; ++k , ++i ){
        result[i] = lc->rings_g[idxList[k]];
        dim_col[i] = lc->col_ring_size[idxList[k]];
    }
    return index+1;
}

int host_get_ring_match_data(int *dim_col, int **result, int *idxList, int *elems, int elem_size, GpuLabelClass *lc){
    int index;

    for( int i = 0; i < elem_size ; ++i){
        for( int j = 0 ; j < lc->g_size ; ++j ){
            if( lc->g[j] == elems[i] ) idxList[index] = index;
            index++;
        }
    }
    for ( int k = 0 , i = 0 ; k < index+1 ; ++k , ++i ){
        result[i] = lc->rings_g[idxList[k]];
        dim_col[i] = lc->col_ring_size[idxList[k]];
    }
    return index+1;
}
// function used in matchable, returns a 1D array containing the ring data related to vertex v
// returns the size of the 1D array
__device__
int kernel_matchable_ring_data(int *result, int v, GpuLabelClass *lc){
    int index=0;
    for ( int i = 0 ; i < lc->g_size ; ++i){
        if ( lc->g[i] == v ) result[index] = index;
        index++;
    }
    return index;
}


int host_matchable_ring_data(int *result, int v, GpuLabelClass *lc){
    int index=0;
    for ( int i = 0 ; i < lc->g_size ; ++i){
        if ( lc->g[i] == v ) result[index] = index;
        index++;
    }
    return index;
}

// return the best select label given an array of labels
__device__
void kernel_select_label(GpuLabelClass *label , GpuLabelClass *lcs, int map_size, int lcs_size){
    int min = 999;
    int max = 0;
    for( int i = 0 ; i < lcs_size ; ++i ){
        if( lcs[i].adj == 1 || map_size == 0 ){
            if( lcs[i].g_size > lcs[i].h_size ) max = lcs[i].g_size;
            else max = lcs[i].h_size;
            if( max < min ){
                min = max;
                *label = lcs[i];
            }
        }
    }
    return;
}

// compute the bound given a 1D array of struct GpuLabelClass and its size
__device__
int calc_bound(GpuLabelClass *lcs, int lc_size) {
    int bound = 0;
    for( int i = 0 ; i < lc_size ; ++i){
        if ( lcs[i].g_size > lcs[i].h_size ) bound = bound + lcs[i].h_size;
        else bound = bound + lcs[i].g_size;
    }
    return bound;
}


//return = size of the friends
//friend is the OUTPUT
__device__
int hoodG(int *friends,int vtx, float edge, float **g0) {
    int size = 0;
    for (int i = 0; i < size_gpu_g0_row; i++) {
        if (g0[i][vtx] == edge && vtx != i) {
            friends[size] = i;
            size++;
        }
    }
    return size;
}


int host_hoodG(int *friends,int vtx, float edge, float **g0) {
    int size = 0;
    for (int i = 0; i < size_gpu_g0_row; i++) {
        if (g0[i][vtx] == edge && vtx != i) {
            friends[size] = i;
            size++;
        }
    }
    return size;
}

// result == size of generated label
// output is l_draft
// input : v
__device__
int kernel_gen_new_Labels(GpuLabelClass *l_draft ,  int v, int w, GpuLabelClass *lcs, int lcs_size, int *v_conn, int *w_conn, int **v_c_rings, int *friends, int *idxList, int *dim_col) {
    int vs,ws, draft_size = 0;
    int dim_row;
    for ( int i = 0 ; i < lcs_size ; ++i ){
        for ( int j = 0 ; j < size_edge_labels ; ++j ){
            int friendsize = hoodG(friends, v, gpu_edge_labels[j], gpu_g0 );
            for ( int k = 0 , vs = 0 ; k < friendsize ; ++k ){
                if( contains(friends[k], lcs[i].g, lcs[i].g_size) ){ v_conn[vs] = friends[k]; vs++; }
            }

            dim_row = kernel_get_ring_match_data(dim_col, v_c_rings, idxList ,v_conn, vs+1, &lcs[i] );

            friendsize = hoodG(friends, w, gpu_edge_labels[j], gpu_g1 );
            for ( int k = 0 , ws = 0 ; k < friendsize ; ++k ){
                if( contains(friends[k], lcs[i].h, lcs[i].h_size) ){ w_conn[ws] = friends[k]; ws++; }
            }

            int adj;
            if ( ws > 0 && vs > 0 ){
                if( gpu_edge_labels[j] != 0.0 || lcs[i].adj == 1 ) adj = 1;
                else adj = 0;
                l_draft[draft_size].g_size = vs;
                l_draft[draft_size].h_size = ws;
                l_draft[draft_size].row_ring_size = dim_row;
                l_draft[draft_size].col_ring_size = dim_col;
                l_draft[draft_size].g = v_conn;
                l_draft[draft_size].h = w_conn;
                l_draft[draft_size].adj = adj;
                for( int c = 0 ; c < 4 ; c++){
                    l_draft[draft_size].label[c] = lcs[i].label[c];
                }
                l_draft[draft_size].rings_g = v_c_rings;
                draft_size++;
            }
        }
    }
    return draft_size;
}


int host_gen_new_Labels(GpuLabelClass *l_draft,int v, int w, GpuLabelClass *lcs, int lcs_size, int *v_conn, int *w_conn, int **v_c_rings, int *friends, int *idxList, int *dim_col) {
    int vs,ws, draft_size = 0;
    int dim_row;
    for ( int i = 0 ; i < lcs_size ; ++i ){
        for ( int j = 0 ; j < size_edge_labels ; ++j ){
            int friendsize = host_hoodG(friends, v, gpu_edge_labels[j], gpu_g0 );
            for ( int k = 0 , vs = 0 ; k < friendsize ; ++k ){
                if( host_contains(friends[k], lcs[i].g, lcs[i].g_size) ){ v_conn[vs] = friends[k]; vs++; }
            }

            dim_row = host_get_ring_match_data(dim_col, v_c_rings, idxList ,v_conn, vs+1, &lcs[i] );

            friendsize = host_hoodG(friends, w, gpu_edge_labels[j], gpu_g1 );
            for ( int k = 0 , ws = 0 ; k < friendsize ; ++k ){
                if( host_contains(friends[k], lcs[i].h, lcs[i].h_size) ){ w_conn[ws] = friends[k]; ws++; }
            }

            int adj;
            if ( ws > 0 && vs > 0 ){
                if( gpu_edge_labels[j] != 0.0 || lcs[i].adj == 1 ) adj = 1;
                else adj = 0;
                l_draft[draft_size].g_size = vs;
                l_draft[draft_size].h_size = ws;
                l_draft[draft_size].row_ring_size = dim_row;
                l_draft[draft_size].col_ring_size = dim_col;
                l_draft[draft_size].g = v_conn;
                l_draft[draft_size].h = w_conn;
                l_draft[draft_size].adj = adj;
                for( int c = 0 ; c < 4 ; c++){
                    l_draft[draft_size].label[c] = lcs[i].label[c];
                }
                l_draft[draft_size].rings_g = v_c_rings;
                draft_size++;
            }
        }
    }
    return draft_size;
}

// ------ FRA functions //


//helper function
void vectorToPointerEdge(float *gpu_edge_labels){
    if(edge_labels.size() == 0){
        gpu_edge_labels = nullptr;
        return;
    }
    int size = 0;
    for(float edg : edge_labels){
        gpu_edge_labels[size] = edg;
        size++;
    }

    return;
}


void vectorToPointerMatrix(const std::vector<std::vector<float>>& g,
                           float** gpu_g
) {
    // Get dimensions of the vector
    int numRows = g.size();
    if (numRows == 0) {
        // Empty vector, set pointers to nullptr
        gpu_g = nullptr;
        return;
    }

    int numCol = g[0].size();

    for (int i = 0; i < numRows; ++i) {
        for (int j = 0; j < numCol; ++j) {
            gpu_g[i][j] = g[i][j];
        }
    }
}


void LabelFromCpuToGpu(GpuLabelClass *new_label, const vector<LabelClass>& old_label ){

    for (int idx = 0 ; idx < old_label.size() ; ++idx ){
        new_label[idx].g_size = old_label.at(idx).g.size();
        new_label[idx].h_size = old_label.at(idx).h.size();
        new_label[idx].row_ring_size = old_label.at(idx).rings_g.size();
        new_label[idx].adj = old_label.at(idx).adj;
        strcpy(new_label[idx].label, old_label.at(idx).label.c_str() );

        for ( int j = 0 ; j < old_label.at(idx).g.size() ; ++j ){
            new_label[idx].g[j] = old_label.at(idx).g.at(j);
        }
        for ( int j = 0 ; j < old_label.at(idx).h.size() ; ++j ){
            new_label[idx].h[j] = old_label.at(idx).h.at(j);}

        for ( int row = 0 ; row < old_label.at(idx).rings_g.size() ; ++row ){
            new_label[idx].col_ring_size[row] = old_label.at(idx).rings_g.at(row).size();
            for ( int col = 0 ; col < old_label.at(idx).rings_g.at(row).size() ; ++col ){
                new_label[idx].rings_g[row][col] = old_label.at(idx).rings_g.at(row).at(col);
            }
        }

    }
}


__device__ bool kernel_matchable(int*  v_ring_atoms, int v, int w, GpuLabelClass *lc) {
    int size  = kernel_matchable_ring_data(v_ring_atoms, v, lc );
    if( size > 0  ) {
        for(int i = 0; i< size; i++){
            if( v_ring_atoms[i] == -1 )return false;
            if( v_ring_atoms[i] == w ) return true;
        }
        return false;
    }
    return true;
}

bool host_matchable(int*  v_ring_atoms, int v, int w, GpuLabelClass *lc) {
    int size  = host_matchable_ring_data(v_ring_atoms, v, lc );
    if( size > 0  ) {
        for(int i = 0; i< size; i++){
            if( v_ring_atoms[i] == -1 )return false;
            if( v_ring_atoms[i] == w ) return true;
        }
        return false;
    }
    return true;
}



// vtx_set: selected label class
// g: selected graph
__device__ void kernel_select_vertex(int *result, int *result_pos, int *vtx_set, int vtx_size, float **g, int num_row, int num_column) {
    int max_deg = -1;
    int vtx = 0;

    for(int i = 0; i < vtx_size; i++){
        int deg = 0;
        for(int j = 0; j < num_column; j++){
            int consider = g[vtx_set[i]][j];
            if(consider != 0){
                deg++;
            }
        }

        if(deg>max_deg){
            max_deg = deg;
            *result = vtx_set[i];
            *result_pos = i;
        }
    }

}


// vtx_set: selected label class
// g: selected graph
int host_select_vertex(int result, int *result_pos, int *vtx_set, int vtx_size, float **g, int num_row, int num_column) {
    int max_deg = -1;
    int vtx = 0;

    for(int i = 0; i < vtx_size; i++){
        int deg = 0;
        for(int j = 0; j < num_column; j++){
            int consider = g[vtx_set[i]][j];
            if(consider != 0){
                deg++;
            }
        }

        if(deg>max_deg){
            max_deg = deg;
            result = vtx_set[i];
            *result_pos = i;
        }
    }
    return result;
}







size_t calcSize(const vector<LabelClass>& lcs) {
    size_t result=0;
    for( const LabelClass& lc : lcs) {
        result = result + (lc.g.size()+lc.h.size());
        for( vector<int> i : lc.rings_g ) {
            result += i.size();
        }
        result += 3;
    }
    return result;
}

void printLabelClass(const LabelClass& lb) {
        cout<< lb.label << " [ ";
        cout<< " G("<< lb.g.size() << "): ";
        if(!lb.g.empty()) {for ( int i : lb.g ) cout<<"["<<i<<"]";}
        cout<< " H("<< lb.h.size() << "): ";
        if(!lb.h.empty()) {for ( int i : lb.h ) cout<<"["<<i<<"]";}
        cout<< " RINGS("<< lb.rings_g.size() << "): [";
        for( vector<int> i : lb.rings_g ){cout<<"("<<i.size()<<")"<<"["; for( int j: i ) cout<<j<<", ";  cout<<" ]";}
        cout<<"]";
        cout<< " edge : " <<lb.adj<<" " ;
        cout<< lb.label << " ] "<<endl;
}




LabelClass *select_label(std::vector<LabelClass*>& label_classes, int map_size);

bool matchable(const int v,const int w, LabelClass lc ) {
    std::vector<int> vector;
    vector.push_back(v);
    std::vector<int>  v_ring_atoms = {};
    v_ring_atoms = lc.get_ring_match_data(vector).at(0);

    if( !v_ring_atoms.empty() ) {
        for(const int x : v_ring_atoms){
            if( x == -1 )return false;
            if( x == w ) return true;}
        return false;
    }
    return true;
}

vector<LabelClass> genNewLabels(int v, int w, const vector<LabelClass>& lcs) {
    vector<LabelClass> l_draft;

    for(LabelClass label : lcs){
        for(float edge_l : edge_labels){
            std::vector<int> v_conn;
            std::vector<int> w_conn;
            std::vector<std::vector<int> > v_c_rings;

            for(int vtx : hood(v,g0,edge_l)){if( std::find(label.g.begin(),label.g.end(),vtx) != label.g.end() )   v_conn.push_back(vtx);}

            v_c_rings = label.get_ring_match_data(v_conn);

            for(int vtx : hood(w,g1,edge_l)){if(std::find(label.h.begin(),label.h.end(),vtx) != label.h.end() )  w_conn.push_back(vtx);}

            int adj;
            if(!v_conn.empty() && !w_conn.empty()){
                if(edge_l != 0.0 || label.adj == 1) adj = 1;
                else adj = 0;
                LabelClass tmp(v_conn,w_conn,v_c_rings,adj, label.label);
                l_draft.push_back(tmp);
            }
        }
    }
    return l_draft;
}




bool solve_mcs() {

    /*queue_elem elem =  Q.back();

    Q.pop_back();

    vector<LabelClass> lcs = elem.labels;
    vector<pair<int,int >> m_local = elem.m_local;

    std::vector<LabelClass*> label_class_pointers;
    label_class_pointers.reserve(lcs.size());
    for (LabelClass& item : lcs) {label_class_pointers.push_back(&item);}


    LabelClass *lcc = select_label(label_class_pointers, m_local.size());
    if ( m_local.size() + calc_bound(lcs) <= m_best.size() || ( !lcc && !m_local.empty() )  ){ if( !Q.empty() ){ return true; } return false;}

    queue_elem qel;
    LabelClass lc = *lcc;
    pair<int,int> m_temp;

    for( int v : lc.g )  {
        for ( int w : lc.h ) {
            if ( !matchable(v,w,lc) ) continue;
            m_temp.first = v;
            m_temp.second = w;
            m_local.push_back(m_temp);
            qel.labels = genNewLabels(v,w,lcs);
            qel.m_local = m_local;
            Q.push_back(qel);
            if ( m_local.size() > m_best.size() ) m_best = m_local;
            m_local.pop_back();
        }
    }

    return true;*/
}






vector<pair<int,int>> gpu_mc_split(const std::vector<std::vector<float>>& g00, const std::vector<std::vector<float>>& g11,
                                   const std::vector<std::string>& l0, const std::vector<std::string>& l1,
                                   std::vector<std::vector<int> >& ring_classes) {
    cout << "CPU: Initializing ..." << endl;
    //vars
    g0 = g00;
    g1 = g11;
    ThreadVar *thread_pool;
    size_t N = DIM_POOL * DIM_POOL;
    edge_labels = cpu_gen_bond_labels(g0, g1);
    int min_mol_size = std::min(l0.size(), l1.size());
    std::vector <LabelClass> initial_label_classes = cpu_gen_initial_labels(l0, l1, ring_classes);
    GpuLabelClass **gpu_initial_label_classes;
    int gpu_initial_label_classes_size = initial_label_classes.size();
    Pair m_local;
    vector <LabelClass> lcs;

    //cuda Mallocs
    //cuda malloc edge labels
    hipMallocManaged(&gpu_edge_labels, sizeof(float) * edge_labels.size());
    //cuda malloc adj matrix mol 0
    hipMallocManaged((void **) &gpu_g0, l0.size() * sizeof(float *));
    for (int i = 0; i < l0.size(); ++i) { hipMallocManaged((void **) &(gpu_g0[i]), l0.size() * sizeof(float)); }
    //cuda malloc adj matrix mol 1
    hipMallocManaged((void **) &gpu_g1, l1.size() * sizeof(float *));
    for (int i = 0; i < l1.size(); ++i) { hipMallocManaged((void **) &(gpu_g1[i]), l1.size() * sizeof(float)); }
    //cuda malloc GpuLabelClass array
    int size = initial_label_classes.size() * 2;
    hipMallocManaged(&gpu_initial_label_classes, initial_label_classes.size() * sizeof(GpuLabelClass *));
    for (int k = 0; k < initial_label_classes.size(); ++k) {
        hipMallocManaged(&gpu_initial_label_classes[k], sizeof(GpuLabelClass) * size);
        for (int i = 0; i < size; i++) {
            hipMallocManaged(&gpu_initial_label_classes[k][i].col_ring_size, sizeof(int *) * min_mol_size);
            hipMallocManaged(&gpu_initial_label_classes[k][i].g, sizeof(int) * l0.size());
            hipMallocManaged(&gpu_initial_label_classes[k][i].h, sizeof(int) * l1.size());
            hipMallocManaged(&gpu_initial_label_classes[k][i].rings_g, sizeof(int *) * min_mol_size);
            for (int j = 0; j < l0.size(); ++j) {
                hipMallocManaged(&(gpu_initial_label_classes[k][i].rings_g[j]), l0.size() * sizeof(int));}}}
    //cudamalloc / initialize pool
    hipMallocManaged(&thread_pool, sizeof(ThreadVar) * N);
    for (int j = 0; j < N; ++j) {
        thread_pool->labels_size = 0;
        thread_pool->m_size = 0;
        hipMallocManaged(&thread_pool[j].labels, sizeof(gpu_initial_label_classes));
        hipMallocManaged(&thread_pool[j].m_local, sizeof(Pair) * min_mol_size);
    }

    //initialize
    //init edge labels
    vectorToPointerEdge(gpu_edge_labels);
    size_edge_labels = edge_labels.size();
    //init adj matrix mol0
    vectorToPointerMatrix(g0, gpu_g0);
    size_gpu_g0_row = g0.size();
    size_gpu_g0_col = g0[0].size();
    //init adj matrix mol 1
    vectorToPointerMatrix(g1, gpu_g1);
    size_gpu_g1_row = g1.size();
    size_gpu_g1_col = g1[0].size();

    cout << "CPU: Initializing thread pool" << endl;


    int v,w,n_threads=0;
    for( LabelClass lc : initial_label_classes ) {
        v = select_vertex(lc.g,g0);
        w = select_vertex(lc.h,g1);
        if( !matchable(v,w,lc ) ) continue;
        m_local.first = v;
        m_local.second = w;
        lcs = genNewLabels(v,w,initial_label_classes);
        LabelFromCpuToGpu(gpu_initial_label_classes[n_threads],lcs);
        thread_pool[n_threads].labels = gpu_initial_label_classes[n_threads];
        thread_pool[n_threads].labels_size = lcs.size();
        thread_pool[n_threads].m_size = 1;
        thread_pool[n_threads].m_local[0] = m_local;
        n_threads++;
    }



    for ( int j = 0 ; j < n_threads ; ++j ){
        for ( int k = 0 ; k < thread_pool[j].labels_size ; ++k ){
            cout<< "\nLABEL : "<<thread_pool[j].labels[k].label;
            cout<< "\nADJ : "<<thread_pool[j].labels[k].adj;
            cout<<"\n G :  ";
            for ( int row = 0 ; row < thread_pool[j].labels[k].g_size ; row++ ){
                cout<<"[ "<< thread_pool[j].labels[k].g[row]<<" ]";
            }
            cout<<"\n H :  ";
            for ( int row = 0 ; row < thread_pool[j].labels[k].h_size ; row++ ){
                cout<<"[ "<< thread_pool[j].labels[k].h[row]<<" ]";
            }
            cout<<"\n ALL RINGS :  ";
            for ( int row = 0 ; row < thread_pool[j].labels[k].row_ring_size ; row++ ){
                cout<<"\n{{ ";
                for ( int col = 0 ; col < thread_pool[j].labels[k].col_ring_size[row] ; col++ ){
                    cout<<"[ "<< thread_pool[j].labels[k].rings_g[row][col] <<" ]";
                }
                cout<<" }} ";
            }
        }
    }



    /*bool flag;
    do{flag = solve_mcs();}while(flag);*/




    //hipFree
    hipFree( gpu_edge_labels );
    for (int i = 0; i < l0.size(); ++i) {hipFree(gpu_g0[i]);}
    hipFree(gpu_g0);
    for (int i = 0; i < l1.size(); ++i) {hipFree(gpu_g1[i]);}
    hipFree(gpu_g1);
    for ( int k = 0; k < initial_label_classes.size() ; ++k ) {
        for (int i = 0; i < size; i++) {
            hipFree(gpu_initial_label_classes[k][i].col_ring_size);
            hipFree(gpu_initial_label_classes[k][i].g);
            hipFree(gpu_initial_label_classes[k][i].h);
            for (int j = 0; j < l0.size(); ++j) {
                hipFree(gpu_initial_label_classes[k][i].rings_g[j]);
            }
            hipFree(gpu_initial_label_classes[k][i].rings_g);
        }
        hipFree(gpu_initial_label_classes[k]);
    }
    hipFree( gpu_initial_label_classes);


    return m_best;
}