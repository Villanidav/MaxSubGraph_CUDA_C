#include "hip/hip_runtime.h"
//
// Created by davide on 5/3/24.
//

#include "test.hpp"
#include <vector>
#include <string>
#include <queue>
#include <algorithm>
#include <unordered_set>

using namespace std;
const int DIM_POOL = 16;
std::vector<std::vector<float>> g0;
std::vector<std::vector<float>> g1;
std::vector<float> edge_labels;
vector<pair<int,int>> m_best;

__device__ float *gpu_edge_labels;
__device__ int size_edge_labels;

__device__ float **gpu_g0;
__device__ int size_gpu_g0_row;
__device__ int size_gpu_g0_col;

__device__ float **gpu_g1;
__device__ int size_gpu_g1_row;
__device__ int size_gpu_g1_col;

typedef struct{
    int g_size;
    int h_size;
    int row_ring_size;
    int *col_ring_size;
    int *g;
    int *h;
    int adj;
    char label[4];
    int **rings_g; 
}GpuLabelClass;


typedef struct{
    int first;
    int second;
}Pair;


typedef struct {
    int labels_size;
    int m_size;
    GpuLabelClass *labels;
    Pair *m_local;
}ThreadVar;

// vtx_set: selected label class
// g: selected graph
int select_vertex(std::vector<int>& vtx_set, std::vector<std::vector<float> >& g) {
    // selects node from graph given a label, choosing an adjacent node with the maximum degree

    int max_deg = -1;
    int vtx = 0;
    for (int c_vtx : vtx_set) {
        int deg = 0;
        for (float i : g[c_vtx]) {
            if (i != 0) {
                deg += 1;
            }
        }

        if (deg > max_deg) {
            max_deg = deg;
            vtx = c_vtx;
        }
    }
    return vtx;
}
std::vector<std::string> find_common_strings(const std::vector<std::string>& l0, const std::vector<std::string>& l1) {
    // Utilizzare un set per memorizzare ed effettuare velocemente la ricerca di stringhe comuni uniche
    std::unordered_set<std::string> common_strings(l0.begin(), l0.end());

    // Vettore per memorizzare le stringhe comuni trovate
    std::vector<std::string> result;

    // Trovare le intersezioni tra le stringhe della seconda lista e le stringhe nel set
    for (const std::string& str : l1) {
        // Se la stringa è presente nel set delle stringhe comuni
        if (common_strings.find(str) != common_strings.end()) {
            // Aggiungila al risultato
            result.push_back(str);
            // Rimuovi la stringa dal set per evitare duplicati
            common_strings.erase(str);
        }
    }
    return result;
}
std::vector<LabelClass> cpu_gen_initial_labels(const std::vector<std::string>& l0, const std::vector<std::string>& l1,     std::vector<std::vector<int> >& ring_classes){
    std::vector<LabelClass> label_classes;
    const std::vector<string> common_labels = find_common_strings(l0,l1);


    for (const std::string& label : common_labels) {
        // Filter atoms and ring data based on label
        std::vector<int> g_elems;
        std::vector<std::vector<int> > g_ring_classes;
        for (size_t i = 0; i < l0.size(); ++i) {
            if (l0[i] == label) {
                g_elems.push_back(i);
                if( !ring_classes.empty() ) g_ring_classes.push_back(ring_classes[i]); // Assuming ring_classes access by index

            }
        }

        std::vector<int> h_elems;
        for (size_t j = 0; j < l1.size(); ++j) {
            if (l1[j] == label) {
                h_elems.push_back(j);
            }
        }

        LabelClass label_tmp(g_elems,h_elems,g_ring_classes,0, label);
        label_classes.push_back(label_tmp);

    }
    return label_classes;
}
std::vector<float> cpu_gen_bond_labels(const std::vector<std::vector<float> >& g0, const std::vector<std::vector<float> >& g1) {

    std::vector<float> all_labels;
    all_labels.push_back(0.0);
    all_labels.push_back(1.0);
    all_labels.push_back(1.5);
    all_labels.push_back(2.0);
    all_labels.push_back(3.0);
    all_labels.push_back(4.0);
    all_labels.push_back(5.0);
    all_labels.push_back(6.0);
    // Vector to store potential bond labels (extracted from both matrices)
    std::vector<float> intersection;

    // Iterate over rows and columns of g0
    for (size_t i = 0; i < g0.size(); ++i) {
        for (size_t j = 0; j < g0[i].size(); ++j) {
            float current_label = g0[i][j];

            // Check if the label exists in each row of g1 (avoid nested loops)
            bool found_in_g1 = false;
            for (const std::vector<float>& row : g1) {
                if (std::find(row.begin(), row.end(), current_label) != row.end()) {
                    found_in_g1 = true;
                    break;
                }
            }

            // If found in g1 and not already in the intersection, add it
            if (found_in_g1 && (intersection.empty() || intersection.back() != current_label)) {
                intersection.push_back(current_label);
            }
        }
    }


    // Sort the intersection vector for desired output
    std::sort(intersection.begin(), intersection.end());
    // Use unique to remove consecutive duplicates (may leave gaps)
    intersection.erase(std::unique(intersection.begin(), intersection.end()), intersection.end());

    // Resize the vector to remove empty space from erasing (optional)
    intersection.resize(intersection.size());

    // Return the intersection vector containing common bond labels (sorted and unique)
    return intersection;
}
std::vector<int> hood(int vtx, const std::vector<std::vector<float>>& g, float edge) {
    // Return the neighbors of a specified node, with the specified bond type.
    std::vector<int> friends;
    for (std::size_t i = 0; i < g.size(); ++i) {
        if (g[i][vtx] == edge && static_cast<std::size_t>(vtx) != i) {
            friends.push_back(i);
        }
    }

    return friends;
}




__device__
bool contains(int value, int *arr, int size) {
    for (int i = 0; i < size; ++i) {
        if (arr[i] == value) {
            return true;
        }
    }
    return false;
}


//      puts into a 2D array the data regarding indexes of rings related to the array of elements
//      2D-array that will contain the result that will be modified
//      1D-array containing idxList
//      1D-array of elements
//      int size of elements
__device__
int kernel_get_ring_match_data(int *dim_col, int **result, int *idxList, int *elems, int elem_size, GpuLabelClass *lc){
    int index;

    for( int i = 0; i < elem_size ; ++i){
        for( int j = 0 ; j < lc->g_size ; ++j ){
            if( lc->g[j] == elems[i] ) idxList[index] = index;
            index++;
        }
    }
    for ( int k = 0 , i = 0 ; k < index+1 ; ++k , ++i ){
        result[i] = lc->rings_g[idxList[k]];
        dim_col[i] = lc->col_ring_size[idxList[k]];
    }
    return index+1;
}

// function used in matchable, returns a 1D array containing the ring data related to vertex v
// returns the size of the 1D array 
__device__
int kernel_matchable_ring_data(int *result, int v, GpuLabelClass *lc){
    int index=0;
    for ( int i = 0 ; i < lc->g_size ; ++i){
        if ( lc->g[i] == v ) result[index] = index;
        index++;
    }
    return index;
}

// return the best select label given an array of labels
__device__
void kernel_select_label(GpuLabelClass *label , GpuLabelClass *lcs, int map_size, int lcs_size){
    int min = 999;
    int max = 0;
    for( int i = 0 ; i < lcs_size ; ++i ){
        if( lcs[i].adj == 1 || map_size == 0 ){
            if( lcs[i].g_size > lcs[i].h_size ) max = lcs[i].g_size;
            else max = lcs[i].h_size;
            if( max < min ){
                min = max;
                *label = lcs[i];
            }
        }
    }
    return;
}

// compute the bound given a 1D array of struct GpuLabelClass and its size
__device__
int calc_bound(GpuLabelClass *lcs, int lc_size) {
    int bound = 0;
    for( int i = 0 ; i < lc_size ; ++i){
        if ( lcs[i].g_size > lcs[i].h_size ) bound = bound + lcs[i].h_size;
        else bound = bound + lcs[i].g_size;
    }
    return bound;
}


//return = size of the friends
//friend is the OUTPUT
__device__
int hoodG(int *friends,int vtx, float edge, float **g0) {
    int size = 0;
    for (int i = 0; i < size_gpu_g0_row; i++) {
        if (g0[i][vtx] == edge && vtx != i) {
            friends[size] = i;
            size++;
        }
    }
    return size;
}


// result == size of generated label
// output is l_draft
// input : v
__device__
int kernel_gen_new_Labels(GpuLabelClass *l_draft,int v, int w, GpuLabelClass *lcs, int lcs_size, int *v_conn, int *w_conn, int **v_c_rings, int *friends, int *idxList, int *dim_col) {
    int vs,ws, draft_size = 0;
    int dim_row;
    for ( int i = 0 ; i < lcs_size ; ++i ){
        for ( int j = 0 ; j < size_edge_labels ; ++j ){
            int friendsize = hoodG(friends, v, gpu_edge_labels[j], gpu_g0 );
            for ( int k = 0 , vs = 0 ; k < friendsize ; ++k ){
                if( contains(friends[k], lcs[i].g, lcs[i].g_size) ){ v_conn[vs] = friends[k]; vs++; }
            }

            dim_row = kernel_get_ring_match_data(dim_col, v_c_rings, idxList ,v_conn, vs+1, &lcs[i] );

            friendsize = hoodG(friends, w, gpu_edge_labels[j], gpu_g1 );
            for ( int k = 0 , ws = 0 ; k < friendsize ; ++k ){
                if( contains(friends[k], lcs[i].h, lcs[i].h_size) ){ w_conn[ws] = friends[k]; ws++; }
            }

            int adj;
            if ( ws > 0 && vs > 0 ){
                if( gpu_edge_labels[j] != 0.0 || lcs[i].adj == 1 ) adj = 1;
                else adj = 0;
                l_draft[draft_size].g_size = vs;
                l_draft[draft_size].h_size = ws;
                l_draft[draft_size].row_ring_size = dim_row;
                l_draft[draft_size].col_ring_size = dim_col;
                l_draft[draft_size].g = v_conn;
                l_draft[draft_size].h = w_conn;
                l_draft[draft_size].adj = adj;
                for( int c = 0 ; c < 4 ; c++){
                    l_draft[draft_size].label[c] = lcs[i].label[c];
                }
                l_draft[draft_size].rings_g = v_c_rings;
                draft_size++;
            }
        }
    }
    return draft_size;
}



// ------ FRA functions //


//helper function
void vectorToPointerEdge(float *gpu_edge_labels){
    if(edge_labels.size() == 0){
        gpu_edge_labels = nullptr;
        return;
    }
    int size = 0;
    for(float edg : edge_labels){
        gpu_edge_labels[size] = edg;
        size++;
    }

    return;
}


void vectorToPointerMatrix(const std::vector<std::vector<float>>& g,
                           float** gpu_g
                           ) {
    // Get dimensions of the vector
    int numRows = g.size();
    if (numRows == 0) {
        // Empty vector, set pointers to nullptr
        gpu_g = nullptr;
        return;
    }

    int numCol = g[0].size();

    for (int i = 0; i < numRows; ++i) {
        for (int j = 0; j < numCol; ++j) {
            gpu_g[i][j] = g[i][j];
        }
    }
}


int LabelFromCpuToGpu(GpuLabelClass *new_label, vector<LabelClass>& old_label ){
    if(old_label.size() == 0 ){
        new_label = nullptr;
        return  0;
    }
    int count = 0;
    for(LabelClass lc : old_label){
        int sizeG = 0;
        int sizeH = 0;

        for(int g : lc.g){
            new_label[count].g[sizeG] = g;
            sizeG++;
        }


        for(int h : lc.h){
            new_label[count].h[sizeH] = h;
            sizeH++;
        }


        int row = 0;
        for(vector<int> ring :lc.rings_g ){
            cout<<"inn3n \n";
            int column = 0;
            for(int i : ring){
                cout<<"innn4 \n";
                new_label[count].rings_g[row][column] = i;
                cout<<"innn5\n";
                column++;
            }

            row++;
        }
        
        count++;
        cout<<"in label\n";
    }

return count;
}


__device__ bool kernel_matchable(int*  v_ring_atoms, int v, int w, GpuLabelClass *lc) {
    int size  = kernel_matchable_ring_data(v_ring_atoms, v, lc );
    if( size > 0  ) {
        for(int i = 0; i< size; i++){
            if( v_ring_atoms[i] == -1 )return false;
            if( v_ring_atoms[i] == w ) return true;
            }
        return false;
    }
    return true;
}





size_t calcSize(const vector<LabelClass>& lcs) {
    size_t result=0;
    for( const LabelClass& lc : lcs) {
        result = result + (lc.g.size()+lc.h.size());
        for( vector<int> i : lc.rings_g ) {
            result += i.size();
        }
        result += 3;
    }
    return result;
}

void printLabelClass(const LabelClass& lb) {
    if( true) {
        cout<< lb.label << " [ ";
        cout<< " G("<< lb.g.size() << "): ";
        if(!lb.g.empty()) {for ( int i : lb.g ) cout<<"["<<i<<"]";}
        cout<< " H("<< lb.h.size() << "): ";
        if(!lb.h.empty()) {for ( int i : lb.h ) cout<<"["<<i<<"]";}
        cout<< " RINGS("<< lb.rings_g.size() << "): [";
        for( vector<int> i : lb.rings_g ){cout<<"("<<i.size()<<")"<<"["; for( int j: i ) cout<<j<<", ";  cout<<" ]";}
        cout<<"]";
        cout<< " edge : " <<lb.adj<<" " ;
        cout<< lb.label << " ] "<<endl;
    }
}




LabelClass *select_label(std::vector<LabelClass*>& label_classes, int map_size);

bool matchable(const int v,const int w, LabelClass lc ) {
    std::vector<int> vector;
    vector.push_back(v);
    std::vector<int>  v_ring_atoms = {};
    v_ring_atoms = lc.get_ring_match_data(vector).at(0);

    if( !v_ring_atoms.empty() ) {
        for(const int x : v_ring_atoms){
            if( x == -1 )return false;
            if( x == w ) return true;}
        return false;
    }
    return true;
}

vector<LabelClass> genNewLabels(int v, int w, const vector<LabelClass>& lcs) {
    vector<LabelClass> l_draft;

    for(LabelClass label : lcs){
        for(float edge_l : edge_labels){
            std::vector<int> v_conn;
            std::vector<int> w_conn;
            std::vector<std::vector<int> > v_c_rings;

            for(int vtx : hood(v,g0,edge_l)){if( std::find(label.g.begin(),label.g.end(),vtx) != label.g.end() )   v_conn.push_back(vtx);}

            v_c_rings = label.get_ring_match_data(v_conn);

            for(int vtx : hood(w,g1,edge_l)){if(std::find(label.h.begin(),label.h.end(),vtx) != label.h.end() )  w_conn.push_back(vtx);}

            int adj;
            if(!v_conn.empty() && !w_conn.empty()){
                if(edge_l != 0.0 || label.adj == 1) adj = 1;
                else adj = 0;
                LabelClass tmp(v_conn,w_conn,v_c_rings,adj, label.label);
                l_draft.push_back(tmp);
            }
        }
    }
    return l_draft;
}




bool solve_mcs() {

    /*queue_elem elem =  Q.back();
    
    Q.pop_back();

    vector<LabelClass> lcs = elem.labels;
    vector<pair<int,int >> m_local = elem.m_local;

    std::vector<LabelClass*> label_class_pointers;
    label_class_pointers.reserve(lcs.size());
    for (LabelClass& item : lcs) {label_class_pointers.push_back(&item);}


    LabelClass *lcc = select_label(label_class_pointers, m_local.size());
    if ( m_local.size() + calc_bound(lcs) <= m_best.size() || ( !lcc && !m_local.empty() )  ){ if( !Q.empty() ){ return true; } return false;}

    queue_elem qel;
    LabelClass lc = *lcc;
    pair<int,int> m_temp;

    for( int v : lc.g )  {
        for ( int w : lc.h ) {
            if ( !matchable(v,w,lc) ) continue;
            m_temp.first = v;
            m_temp.second = w;
            m_local.push_back(m_temp);
            qel.labels = genNewLabels(v,w,lcs);
            qel.m_local = m_local;
            Q.push_back(qel);
            if ( m_local.size() > m_best.size() ) m_best = m_local;
            m_local.pop_back();
        }
    }

    return true;*/
}






vector<pair<int,int>> gpu_mc_split(const std::vector<std::vector<float>>& g00, const std::vector<std::vector<float>>& g11,
                                          const std::vector<std::string>& l0, const std::vector<std::string>& l1,
                                          std::vector<std::vector<int> >& ring_classes){
    g0 = g00;
    g1 = g11;
    edge_labels = cpu_gen_bond_labels(g0, g1);
    int min_mol_size = std::min(l0.size(), l1.size());
    std::vector<LabelClass> initial_label_classes = cpu_gen_initial_labels(l0, l1, ring_classes);
    GpuLabelClass *gpu_initial_label_classes;

    //cuda malloc edge labels
    hipMallocManaged( &gpu_edge_labels, sizeof(float ) * edge_labels.size() );
    //cuda malloc adj matrix mol 0
    hipMallocManaged((void**)&gpu_g0, l0.size() * sizeof(float*));
    for (int i = 0; i < l0.size(); ++i) {hipMallocManaged((void**)&(gpu_g0[i]), l0.size() * sizeof(float));}
    //cuda malloc adj matrix mol 1
    hipMallocManaged((void**)&gpu_g1, l1.size() * sizeof(float*));
    for (int i = 0; i < l1.size(); ++i) {hipMallocManaged((void**)&(gpu_g1[i]), l1.size() * sizeof(float));}
    //cuda malloc GpuLabelClass array
    hipMallocManaged( &gpu_initial_label_classes, initial_label_classes.size() * sizeof(GpuLabelClass *));
    for( int i = 0 ; i < initial_label_classes.size() ; i++ ){
        gpu_initial_label_classes[i].g_size = initial_label_classes[i].g.size();
        gpu_initial_label_classes[i].h_size = initial_label_classes[i].h.size();
        gpu_initial_label_classes[i].row_ring_size = initial_label_classes[i].rings_g.size();
        hipMallocManaged( &gpu_initial_label_classes[i].col_ring_size , sizeof(int*) * gpu_initial_label_classes[i].row_ring_size );
        for( int r = 0 ; r <  gpu_initial_label_classes[i].row_ring_size ; ++r ){gpu_initial_label_classes[i].col_ring_size[r] = initial_label_classes[i].rings_g[r].size();}
        gpu_initial_label_classes[i].adj = initial_label_classes[i].adj;
         strcpy(gpu_initial_label_classes[i].label, initial_label_classes[i].label.c_str());
        hipMallocManaged( &gpu_initial_label_classes[i].g , sizeof(int) * gpu_initial_label_classes[i].g_size);
        hipMallocManaged( &gpu_initial_label_classes[i].h , sizeof(int) * gpu_initial_label_classes[i].h_size);
        hipMallocManaged( &gpu_initial_label_classes[i].rings_g , sizeof(int*) * gpu_initial_label_classes[i].row_ring_size);
        for (int j = 0; j < gpu_initial_label_classes[i].row_ring_size ; ++j) {
            hipMallocManaged((void**)&(gpu_initial_label_classes[i].rings_g[j]), gpu_initial_label_classes[i].col_ring_size[j] * sizeof(int));}
    }

    //init edge labels
    vectorToPointerEdge(gpu_edge_labels);
    size_edge_labels = edge_labels.size();
    //init adj matrix mol0
    vectorToPointerMatrix(g0,gpu_g0);
    size_gpu_g0_row = g0.size();
    size_gpu_g0_col = g0[0].size();
    //init adj matrix mol 1
    vectorToPointerMatrix(g1,gpu_g1);
    size_gpu_g1_row = g1.size();
    size_gpu_g1_col = g1[0].size();
    //LabelFromCpuToGpu(gpu_initial_label_classes,initial_label_classes);


    //creating the thread pool with dim N
    ThreadVar *thread_pool;
    size_t N = DIM_POOL*DIM_POOL;
    //cudamalloc / initialize pool
    hipMallocManaged( &thread_pool, sizeof(ThreadVar) * N );
    for ( int j = 0 ; j < N ; ++j ){
        thread_pool->labels_size = 0;
        thread_pool->m_size = 0;
        hipMallocManaged( &thread_pool[j].labels , sizeof(gpu_initial_label_classes) );
        hipMallocManaged( &thread_pool[j].m_local , sizeof(Pair) * min_mol_size );}

    Pair m_local;
    vector<LabelClass> lcs;
    int v,w,n_threads=0;
    for( LabelClass lc : initial_label_classes ) {
        v = select_vertex(lc.g,g0);
        w = select_vertex(lc.h,g1);
        if( !matchable(v,w,lc ) ) continue;
        m_local.first = v;
        m_local.second = w;
        cout<<"thread pool\n";
        lcs = genNewLabels(v,w,initial_label_classes);
        LabelFromCpuToGpu(gpu_initial_label_classes,lcs);
        thread_pool[n_threads].labels = gpu_initial_label_classes;
        thread_pool[n_threads].labels_size = lcs.size();
        thread_pool[n_threads].m_size = 1;
        thread_pool[n_threads].m_local[0] = m_local;
        n_threads++;
    }


    for ( int j = 0 ; j < n_threads ; ++j ){
        cout<<"thread pool "<<j<<endl;
        cout<<thread_pool[j].m_local[0].first;
        cout<<" - "<<thread_pool[j].m_local[0].second<<endl;
        cout<<thread_pool[j].labels_size<<endl;
    }

    /*bool flag;
    do{flag = solve_mcs();}while(flag);*/
    return m_best;
}