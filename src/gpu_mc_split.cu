#include "hip/hip_runtime.h"
//
// Created by davide on 5/3/24.
//
#include "test.hpp"
#include <vector>
#include <string.h>
#include <string>
#include <queue>
#include <algorithm>
#include <unordered_set>

using namespace std;
const int DIM_POOL = 8;
std::vector<std::vector<float>> g0;
std::vector<std::vector<float>> g1;
std::vector<float> edge_labels;
//vector<pair<int,int>> m_best;

__device__ float *gpu_edge_labels;
__device__ int size_edge_labels;


__device__ float **gpu_g0;
__device__ int size_gpu_g0_row;
__device__ int size_gpu_g0_col;

__device__ float **gpu_g1;
__device__ int size_gpu_g1_row;
__device__ int size_gpu_g1_col;

typedef struct{
    int g_size;
    int h_size;
    int row_ring_size;
    int *col_ring_size;
    int *g;
    int *h;
    int adj;
    char label[4];
    int **rings_g;
}GpuLabelClass;


typedef struct{
    int first;
    int second;
}Pair;


typedef struct {
    int labels_size;
    int m_size;
    GpuLabelClass *labels;
    GpuLabelClass single_label;
    Pair *m_local;
}ThreadVar;


__shared__ Pair *m_best;
__shared__ int m_best_size;

LabelClass *select_label_DC(std::vector<LabelClass*>& label_classes, int map_size) {

<<<<<<< HEAD


void copyIntArray(int *a, int *b, int sizeb){
    for ( int i = 0 ; i < sizeb ; i++){
        a[i] = b[i];
    }
}


void copyIntMatrix(int **a, int **b, int rowsize, int *colsize )
{
    for( int i = 0 ; i < rowsize ; i++){
        for( int j = 0 ; j < colsize[i] ; j++){
            a[i][j] = b[i][j];
        }
    }
=======
    
    int min_size = 999;
    LabelClass* label = nullptr;

    for ( LabelClass* c_label : label_classes) {
        if (c_label->adj == 1 || map_size == 0) {
            int c_max_size = std::max(c_label->g.size(), c_label->h.size());
            if (c_max_size < min_size) {
                min_size = c_max_size;
                label = c_label;
            }
        }
    }

    return label;
>>>>>>> origin/main
}
// vtx_set: selected label class
// g: selected graph
int select_vertex(std::vector<int>& vtx_set, std::vector<std::vector<float> >& g) {
    // selects node from graph given a label, choosing an adjacent node with the maximum degree

    int max_deg = -1;
    int vtx = 0;
    for (int c_vtx : vtx_set) {
        int deg = 0;
        for (float i : g[c_vtx]) {
            if (i != 0) {
                deg += 1;
            }
        }

        if (deg > max_deg) {
            max_deg = deg;
            vtx = c_vtx;
        }
    }
    return vtx;
}
std::vector<std::string> find_common_strings(const std::vector<std::string>& l0, const std::vector<std::string>& l1) {
    // Utilizzare un set per memorizzare ed effettuare velocemente la ricerca di stringhe comuni uniche
    std::unordered_set<std::string> common_strings(l0.begin(), l0.end());

    // Vettore per memorizzare le stringhe comuni trovate
    std::vector<std::string> result;

    // Trovare le intersezioni tra le stringhe della seconda lista e le stringhe nel set
    for (const std::string& str : l1) {
        // Se la stringa è presente nel set delle stringhe comuni
        if (common_strings.find(str) != common_strings.end()) {
            // Aggiungila al risultato
            result.push_back(str);
            // Rimuovi la stringa dal set per evitare duplicati
            common_strings.erase(str);
        }
    }
    return result;
}
std::vector<LabelClass> cpu_gen_initial_labels(const std::vector<std::string>& l0, const std::vector<std::string>& l1,     std::vector<std::vector<int> >& ring_classes){
    std::vector<LabelClass> label_classes;
    const std::vector<string> common_labels = find_common_strings(l0,l1);


    for (const std::string& label : common_labels) {
        // Filter atoms and ring data based on label
        std::vector<int> g_elems;
        std::vector<std::vector<int> > g_ring_classes;
        for (size_t i = 0; i < l0.size(); ++i) {
            if (l0[i] == label) {
                g_elems.push_back(i);
                if( !ring_classes.empty() ) g_ring_classes.push_back(ring_classes[i]); // Assuming ring_classes access by index

            }
        }

        std::vector<int> h_elems;
        for (size_t j = 0; j < l1.size(); ++j) {
            if (l1[j] == label) {
                h_elems.push_back(j);
            }
        }

        LabelClass label_tmp(g_elems,h_elems,g_ring_classes,0, label);
        label_classes.push_back(label_tmp);

    }
    return label_classes;
}
std::vector<float> cpu_gen_bond_labels(const std::vector<std::vector<float> >& g0, const std::vector<std::vector<float> >& g1) {

    std::vector<float> all_labels;
    all_labels.push_back(0.0);
    all_labels.push_back(1.0);
    all_labels.push_back(1.5);
    all_labels.push_back(2.0);
    all_labels.push_back(3.0);
    all_labels.push_back(4.0);
    all_labels.push_back(5.0);
    all_labels.push_back(6.0);
    // Vector to store potential bond labels (extracted from both matrices)
    std::vector<float> intersection;

    // Iterate over rows and columns of g0
    for (size_t i = 0; i < g0.size(); ++i) {
        for (size_t j = 0; j < g0[i].size(); ++j) {
            float current_label = g0[i][j];

            // Check if the label exists in each row of g1 (avoid nested loops)
            bool found_in_g1 = false;
            for (const std::vector<float>& row : g1) {
                if (std::find(row.begin(), row.end(), current_label) != row.end()) {
                    found_in_g1 = true;
                    break;
                }
            }

            // If found in g1 and not already in the intersection, add it
            if (found_in_g1 && (intersection.empty() || intersection.back() != current_label)) {
                intersection.push_back(current_label);
            }
        }
    }


    // Sort the intersection vector for desired output
    std::sort(intersection.begin(), intersection.end());
    // Use unique to remove consecutive duplicates (may leave gaps)
    intersection.erase(std::unique(intersection.begin(), intersection.end()), intersection.end());

    // Resize the vector to remove empty space from erasing (optional)
    intersection.resize(intersection.size());

    // Return the intersection vector containing common bond labels (sorted and unique)
    return intersection;
}
std::vector<int> hood(int vtx, const std::vector<std::vector<float>>& g, float edge) {
    // Return the neighbors of a specified node, with the specified bond type.
    std::vector<int> friends;
    for (std::size_t i = 0; i < g.size(); ++i) {
        if (g[i][vtx] == edge && static_cast<std::size_t>(vtx) != i) {
            friends.push_back(i);
        }
    }

    return friends;
}




__device__
bool contains(int value, int *arr, int size) {
    for (int i = 0; i < size; ++i) {
        if (arr[i] == value) {
            return true;
        }
    }
    return false;
}


bool host_contains(int value, int *arr, int size) {
    for (int i = 0; i < size; ++i) {
        if (arr[i] == value) {
            return true;
        }
    }
    return false;
}

//      puts into a 2D array the data regarding indexes of rings related to the array of elements
//      2D-array that will contain the result that will be modified
//      1D-array containing idxList
//      1D-array of elements
//      int size of elements
__device__
int kernel_get_ring_match_data(int *dim_col, int **result, int *idxList, int *elems, int elem_size, GpuLabelClass *lc){
    
    int index;
    int k=0;
    for( int i = 0; i < elem_size ; ++i){
        index = 0;
        for( int j = 0 ; j < lc->g_size ; ++j ){
            if( lc->g[j] == elems[i] ) idxList[k] = index;
            index++;
            k++;
        }
    }

    for ( int k = 0 , i = 0 ; k < index ; ++k , ++i ){
        dim_col[i] = lc->col_ring_size[idxList[k]];
        for ( int j = 0 ; j < dim_col[i] ; ++j ){
             result[i][j] = lc->rings_g[idxList[k]][j];
        }
    }
    return index;
}

int host_get_ring_match_data(int *dim_col, int **result, int *idxList, int *elems, int elem_size, GpuLabelClass *lc){
    
    int index;
    int k=0;
    for( int i = 0; i < elem_size ; ++i){
        index = 0;
        for( int j = 0 ; j < lc->g_size ; ++j ){
            if( lc->g[j] == elems[i] ) idxList[k] = index;
            index++;
            k++;
        }
    }

    for ( int k = 0 , i = 0 ; k < index ; ++k , ++i ){
        dim_col[i] = lc->col_ring_size[idxList[k]];
        for ( int j = 0 ; j < dim_col[i] ; ++j ){
             result[i][j] = lc->rings_g[idxList[k]][j];
        }
    }
    return index;
}
// function used in matchable, returns a 1D array containing the ring data related to vertex v
// returns the size of the 1D array
__device__
int kernel_matchable_ring_data(int **result, int v, GpuLabelClass *lc, int *idxList, int *dim_col){
    int index;
    int k=0;
    index = 0;
    for( int j = 0 ; j < lc->g_size ; ++j ){
        if( lc->g[j] == v ) idxList[k] = index;
        index++;
        k++;
    }


    for ( int k = 0 , i = 0 ; k < index ; ++k , ++i ){
        dim_col[i] = lc->col_ring_size[idxList[k]];
        for ( int j = 0 ; j < dim_col[i] ; ++j ){
             result[i][j] = lc->rings_g[idxList[k]][j];
        }
    }
    return index;
}


int host_matchable_ring_data(int *result, int v, GpuLabelClass *lc){
    int index=0;
    for ( int i = 0 ; i < lc->g_size ; ++i){
        if ( lc->g[i] == v ) result[index] = index;
        index++;
    }
    return index;

}

// return the best select label given an array of labels
__device__
void kernel_select_label(GpuLabelClass *label , GpuLabelClass *lcs, int map_size, int lcs_size){
    int min = 999;
    int max = 0;
    for( int i = 0 ; i < lcs_size ; ++i ){
        if( lcs[i].adj == 1 || map_size == 0 ){
            if( lcs[i].g_size > lcs[i].h_size ) max = lcs[i].g_size;
            else max = lcs[i].h_size;
            if( max < min ){
                min = max;
                *label = lcs[i];
            }
        }
    }
    return;
}


void host_select_label(GpuLabelClass *label , GpuLabelClass *lcs, int map_size, int lcs_size){
    int min = 999;
    int max = 0;
    for( int i = 0 ; i < lcs_size ; ++i ){
        if( lcs[i].adj == 1 || map_size == 0 ){
            if( lcs[i].g_size > lcs[i].h_size ) max = lcs[i].g_size;
            else max = lcs[i].h_size;
            if( max < min ){
                min = max;
                *label = lcs[i];
            }
        }
    }
    return;
}

// compute the bound given a 1D array of struct GpuLabelClass and its size
__device__
int kernel_calc_bound(GpuLabelClass *lcs, int lc_size) {
    int bound = 0;
    for( int i = 0 ; i < lc_size ; ++i){
        if ( lcs[i].g_size > lcs[i].h_size ) bound = bound + lcs[i].h_size;
        else bound = bound + lcs[i].g_size;
    }
    return bound;
}


//return = size of the friends
//friend is the OUTPUT
__device__
int hoodG(int *friends,int vtx, float edge, float **g0) {
    int size = 0;
    for (int i = 0; i < size_gpu_g0_row; i++) {
        if (g0[i][vtx] == edge && vtx != i) {
            friends[size] = i;
            size++;
        }
    }
    return size;
}


int host_hoodG(int *friends,int vtx, float edge, float **g0) {
    int size = 0;
    for (int i = 0; i < size_gpu_g0_row; i++) {
        if (g0[i][vtx] == edge && vtx != i) {
            friends[size] = i;
            size++;
        }
    }
    return size;
}

// result == size of generated label
// output is l_draft
// input : v
__device__
void kernel_resize(int *array, int size_arr, int place_availabel){
    int count = 0;
    bool flag = true;
    for(int i = 0; i < size_arr && flag ; ++i){
        if(array[i] == -1){continue;}
        if( i == count){ count ++; continue; }
        array[count] = array[i]; count++;
        if(count == place_availabel){flag = false;}
    }
}

// result == size of generated label
// output is l_draft
// input : v

void host_resize(int *array, int size_arr, int place_availabel){
    int count = 0;
    bool flag = true;
    for(int i = 0; i < size_arr && flag ; ++i){
        if(array[i] == -1){continue;}
        if( i == count){ count ++; continue; }
        array[count] = array[i]; count++;
        if(count == place_availabel){flag = false;}
    }
}


__device__
int kernel_gen_new_Labels(GpuLabelClass *l_draft ,  int v, int w, GpuLabelClass *lcs, int lcs_size, int *idxList) {
    int vs,ws, draft_size = 0;
    int dim_row;
    int count = 0;
    for ( int i = 0 ; i < lcs_size ; ++i ){
        for ( int j = 0 ; j < size_edge_labels ; ++j ){
            int friendsize = hoodG(l_draft[draft_size].g, v, gpu_edge_labels[j], gpu_g0 );
            for ( int k = 0 , vs = 0 ; k < friendsize ; ++k ){
                if( contains(l_draft[draft_size].g[k], lcs[i].g, lcs[i].g_size) ){  vs++;  }
                else{ l_draft[draft_size].g[k] = -1;}
            }
            kernel_resize(l_draft[draft_size].g, friendsize, vs );

            dim_row = kernel_get_ring_match_data(l_draft[draft_size].col_ring_size, l_draft[draft_size].rings_g, idxList ,l_draft[draft_size].g, vs+1, &lcs[i] );

            friendsize = hoodG(l_draft[draft_size].h, w, gpu_edge_labels[j], gpu_g1 );
            for ( int k = 0 , ws = 0 ; k < friendsize ; ++k ){
                if( contains(l_draft[draft_size].h[k], lcs[i].h, lcs[i].h_size) ){  ws++; }
                else {
                    l_draft[draft_size].h[k] = -1;
                }
            }
            kernel_resize(l_draft[draft_size].h, friendsize, ws );

            int adj;
            if ( ws > 0 && vs > 0 ){
                if( gpu_edge_labels[j] != 0.0 || lcs[i].adj == 1 ) adj = 1;
                else adj = 0;
                l_draft[draft_size].g_size = vs;
                l_draft[draft_size].h_size = ws;
                l_draft[draft_size].row_ring_size = dim_row;
                l_draft[draft_size].adj = adj;
                for( int c = 0 ; c < 4 ; c++){
                    l_draft[draft_size].label[c] = lcs[i].label[c];
                }
                
                draft_size++;
            }
        }
    }
    return draft_size;
}


int host_gen_new_Labels(GpuLabelClass *l_draft ,  int v, int w, GpuLabelClass *lcs, int lcs_size, int *idxList) {
    int vs,ws, draft_size = 0;
    int dim_row;
    int count = 0;
    for ( int i = 0 ; i < lcs_size ; ++i ){
        for ( int j = 0 ; j < size_edge_labels ; ++j ){
            int friendsize = host_hoodG(l_draft[draft_size].g, v, gpu_edge_labels[j], gpu_g0 );
            for ( int k = 0 , vs = 0 ; k < friendsize ; ++k ){
                if( host_contains(l_draft[draft_size].g[k], lcs[i].g, lcs[i].g_size) ){  vs++;  }
                else{ l_draft[draft_size].g[k] = -1;}
            }
            host_resize(l_draft[draft_size].g, friendsize, vs );

            dim_row = host_get_ring_match_data(l_draft[draft_size].col_ring_size, l_draft[draft_size].rings_g, idxList ,l_draft[draft_size].g, vs+1, &lcs[i] );

            friendsize = host_hoodG(l_draft[draft_size].h, w, gpu_edge_labels[j], gpu_g1 );
            for ( int k = 0 , ws = 0 ; k < friendsize ; ++k ){
                if( host_contains(l_draft[draft_size].h[k], lcs[i].h, lcs[i].h_size) ){  ws++; }
                else {
                    l_draft[draft_size].h[k] = -1;
                }
            }
            host_resize(l_draft[draft_size].h, friendsize, ws );

            int adj;
            if ( ws > 0 && vs > 0 ){
                if( gpu_edge_labels[j] != 0.0 || lcs[i].adj == 1 ) adj = 1;
                else adj = 0;
                l_draft[draft_size].g_size = vs;
                l_draft[draft_size].h_size = ws;
                l_draft[draft_size].row_ring_size = dim_row;
                l_draft[draft_size].adj = adj;
                for( int c = 0 ; c < 4 ; c++){
                    l_draft[draft_size].label[c] = lcs[i].label[c];
                }
                
                draft_size++;
            }
        }
    }
    return draft_size;
}

// ------ FRA functions //


//helper function
void vectorToPointerEdge(float *gpu_edge_labels){
    if(edge_labels.size() == 0){
        gpu_edge_labels = nullptr;
        return;
    }
    int size = 0;
    for(float edg : edge_labels){
        gpu_edge_labels[size] = edg;
        size++;
    }

    return;
}


void vectorToPointerMatrix(const std::vector<std::vector<float>>& g,
                           float** gpu_g
) {
    // Get dimensions of the vector
    int numRows = g.size();
    if (numRows == 0) {
        // Empty vector, set pointers to nullptr
        gpu_g = nullptr;
        return;
    }

    int numCol = g[0].size();

    for (int i = 0; i < numRows; ++i) {
        for (int j = 0; j < numCol; ++j) {
            gpu_g[i][j] = g[i][j];
        }
    }
}


void LabelFromCpuToGpu(GpuLabelClass *new_label, const vector<LabelClass>& old_label ){

    for (int idx = 0 ; idx < old_label.size() ; ++idx ){
        new_label[idx].g_size = old_label.at(idx).g.size();
        new_label[idx].h_size = old_label.at(idx).h.size();
        new_label[idx].row_ring_size = old_label.at(idx).rings_g.size();
        new_label[idx].adj = old_label.at(idx).adj;
        strcpy(new_label[idx].label, old_label.at(idx).label.c_str() );

        for ( int j = 0 ; j < old_label.at(idx).g.size() ; ++j ){
            new_label[idx].g[j] = old_label.at(idx).g.at(j);
        }
        for ( int j = 0 ; j < old_label.at(idx).h.size() ; ++j ){
            new_label[idx].h[j] = old_label.at(idx).h.at(j);}

        for ( int row = 0 ; row < old_label.at(idx).rings_g.size() ; ++row ){
            new_label[idx].col_ring_size[row] = old_label.at(idx).rings_g.at(row).size();
            for ( int col = 0 ; col < old_label.at(idx).rings_g.at(row).size() ; ++col ){
                new_label[idx].rings_g[row][col] = old_label.at(idx).rings_g.at(row).at(col);
            }
        }

    }
}


__device__ bool kernel_matchable(int**  v_ring_atoms, int v, int w, GpuLabelClass *lc) {
    /*int size  = kernel_matchable_ring_data(v_ring_atoms, v, lc );
    if( size > 0  ) {
        for(int i = 0; i< size; i++){
            if( v_ring_atoms[i] == -1 )return false;
            if( v_ring_atoms[i] == w ) return true;
        }
        return false;
    }*/
    return true;
}

bool host_matchable(int*  v_ring_atoms, int v, int w, GpuLabelClass *lc) {
    int size  = host_matchable_ring_data(v_ring_atoms, v, lc );
    if( size > 0  ) {
        for(int i = 0; i< size; i++){
            if( v_ring_atoms[i] == -1 )return false;
            if( v_ring_atoms[i] == w ) return true;
        }
        return false;
    }
    return true;
}



// vtx_set: selected label class
// g: selected graph
__device__ void kernel_select_vertex(int *result, int *result_pos, int *vtx_set, int vtx_size, float **g, int num_row, int num_column) {
    int max_deg = -1;
    int vtx = 0;

    for(int i = 0; i < vtx_size; i++){
        int deg = 0;
        for(int j = 0; j < num_column; j++){
            int consider = g[vtx_set[i]][j];
            if(consider != 0){
                deg++;
            }
        }

        if(deg>max_deg){
            max_deg = deg;
            *result = vtx_set[i];
            *result_pos = i;
        }
    }

}


// vtx_set: selected label class
// g: selected graph
void host_select_vertex(int *result, int *result_pos, int *vtx_set, int vtx_size, float **g, int num_row, int num_column) {
    int max_deg = -1;
    int vtx = 0;

    for(int i = 0; i < vtx_size; i++){
        int deg = 0;
        for(int j = 0; j < num_column; j++){
            int consider = g[vtx_set[i]][j];
            if(consider != 0){
                deg++;
            }
        }

        if(deg>max_deg){
            max_deg = deg;
            *result = vtx_set[i];
            *result_pos = i;
        }
    }
    return ;
}







size_t calcSize(const vector<LabelClass>& lcs) {
    size_t result=0;
    for( const LabelClass& lc : lcs) {
        result = result + (lc.g.size()+lc.h.size());
        for( vector<int> i : lc.rings_g ) {
            result += i.size();
        }
        result += 3;
    }
    return result;
}

void printLabelClass(const LabelClass& lb) {
        cout<< lb.label << " [ ";
        cout<< " G("<< lb.g.size() << "): ";
        if(!lb.g.empty()) {for ( int i : lb.g ) cout<<"["<<i<<"]";}
        cout<< " H("<< lb.h.size() << "): ";
        if(!lb.h.empty()) {for ( int i : lb.h ) cout<<"["<<i<<"]";}
        cout<< " RINGS("<< lb.rings_g.size() << "): [";
        for( vector<int> i : lb.rings_g ){cout<<"("<<i.size()<<")"<<"["; for( int j: i ) cout<<j<<", ";  cout<<" ]";}
        cout<<"]";
        cout<< " edge : " <<lb.adj<<" " ;
        cout<< lb.label << " ] "<<endl;
}




LabelClass *select_label(std::vector<LabelClass*>& label_classes, int map_size);

bool matchable(const int v,const int w, LabelClass lc ) {
    std::vector<int> vector;
    vector.push_back(v);
    std::vector<int>  v_ring_atoms = {};
    v_ring_atoms = lc.get_ring_match_data(vector).at(0);

    if( !v_ring_atoms.empty() ) {
        for(const int x : v_ring_atoms){
            if( x == -1 )return false;
            if( x == w ) return true;}
        return false;
    }
    return true;
}

vector<LabelClass> genNewLabels(int v, int w, const vector<LabelClass>& lcs) {
    vector<LabelClass> l_draft;

    for(LabelClass label : lcs){
        for(float edge_l : edge_labels){
            std::vector<int> v_conn;
            std::vector<int> w_conn;
            std::vector<std::vector<int> > v_c_rings;

            for(int vtx : hood(v,g0,edge_l)){if( std::find(label.g.begin(),label.g.end(),vtx) != label.g.end() )   v_conn.push_back(vtx);}

            v_c_rings = label.get_ring_match_data(v_conn);

            for(int vtx : hood(w,g1,edge_l)){if(std::find(label.h.begin(),label.h.end(),vtx) != label.h.end() )  w_conn.push_back(vtx);}

            int adj;
            if(!v_conn.empty() && !w_conn.empty()){
                if(edge_l != 0.0 || label.adj == 1) adj = 1;
                else adj = 0;
                LabelClass tmp(v_conn,w_conn,v_c_rings,adj, label.label);
                l_draft.push_back(tmp);
            }
        }
    }
    return l_draft;
}







__global__ 
void kernel_function( ThreadVar *thread_pool_read, ThreadVar *thread_pool_write ){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int space = 4;
    GpuLabelClass *label = nullptr;

    kernel_select_label(label, thread_pool_read[index].labels, 
                        thread_pool_read[index].m_size, thread_pool_read[index].labels_size);

    if ( (thread_pool_read[index].m_size + kernel_calc_bound(thread_pool_read[index].labels, thread_pool_read[index].labels_size  )  
        < m_best_size) || !label  ) return;

    int jump = 0;
    for( int v_idx = 0 ; v_idx < label->g_size ; ++v_idx){
        for( int w_idx = 0 ; w_idx < label->h_size ; ++w_idx){
            if( !kernel_matchable(label->rings_g, label->g[v_idx], label->h[w_idx], label )) continue;

            for( int z = 0 ; z < thread_pool_write[4*index + jump].m_size ; ++z ){
                thread_pool_write[4*index + jump].m_local[z].first = thread_pool_read[index].m_local[z].first;
                thread_pool_write[4*index + jump].m_local[z].second = thread_pool_read[index].m_local[z].second;
            }

            /*kernel_gen_new_labels( thread_pool_write[4*index + jump].labels, label.g[v_idx], label.h[w_idx] ,
                                   thread_pool_read[index].labels , thread_pool_read[index].labels_size ,
                                   );*/

            if( thread_pool_write[4*index + jump].m_size > m_best_size ){
                for( int z = 0 ; z < thread_pool_write[4*index + jump].m_size ; ++z ){
                    m_best[z].first = thread_pool_write[4*index + jump].m_local[z].first;
                    m_best[z].second = thread_pool_write[4*index + jump].m_local[z].second;
                }
            }
        }
    }
}






void cpyThreadPool( ThreadVar *thread_pool_read, ThreadVar *thread_pool_write ){
    int r_idx = 0;
    for( int w_idx = 0; w_idx < DIM_POOL*DIM_POOL ; w_idx++){ 
        thread_pool_read[r_idx].labels_size = thread_pool_write[w_idx].labels_size;
        //da modificare l'if
        if ( thread_pool_write[w_idx].labels_size > 0 ){
            thread_pool_read[r_idx].m_size = thread_pool_write[w_idx].m_size;
            thread_pool_read[r_idx].m_local->first = thread_pool_write[w_idx].m_local->first;
            thread_pool_read[r_idx].m_local->second = thread_pool_write[w_idx].m_local->second;
            for ( int l_idx = 0 ; l_idx < thread_pool_write[w_idx].labels_size ; l_idx++ ){
                thread_pool_read[r_idx].labels[l_idx].row_ring_size = thread_pool_write[w_idx].labels[l_idx].row_ring_size;
                thread_pool_read[r_idx].labels[l_idx].g_size = thread_pool_write[w_idx].labels[l_idx].g_size;
                thread_pool_read[r_idx].labels[l_idx].h_size = thread_pool_write[w_idx].labels[l_idx].h_size;
                copyIntArray( thread_pool_read[r_idx].labels[l_idx].g , thread_pool_write[w_idx].labels[l_idx].g , thread_pool_write[w_idx].labels[l_idx].g_size );
                copyIntArray( thread_pool_read[r_idx].labels[l_idx].h , thread_pool_write[w_idx].labels[l_idx].h , thread_pool_write[w_idx].labels[l_idx].h_size );
                copyIntArray( thread_pool_read[r_idx].labels[l_idx].col_ring_size , thread_pool_write[w_idx].labels[l_idx].col_ring_size , thread_pool_write[w_idx].labels[l_idx].row_ring_size );
                strcpy(thread_pool_write[w_idx].labels[l_idx].label, thread_pool_write[w_idx].labels[l_idx].label );
                copyIntMatrix( thread_pool_write[w_idx].labels[l_idx].rings_g , thread_pool_write[w_idx].labels[l_idx].rings_g, 
                                thread_pool_write[w_idx].labels[l_idx].row_ring_size, thread_pool_write[w_idx].labels[l_idx].col_ring_size );
            }
            r_idx++;
        }
        thread_pool_write[w_idx].labels_size = 0;
    }
}






vector<pair<int,int>> gpu_mc_split(const std::vector<std::vector<float>>& g00, const std::vector<std::vector<float>>& g11,
                                   const std::vector<std::string>& l0, const std::vector<std::string>& l1,
                                   std::vector<std::vector<int> >& ring_classes) {
    //vars
    g0 = g00;
    g1 = g11;
    size_t N = DIM_POOL * DIM_POOL;
    ThreadVar *thread_pool_read;
    ThreadVar *thread_pool_write;
    edge_labels =cpu_gen_bond_labels(g0, g1);
    int min_mol_size = std::min(l0.size(), l1.size());
    std::vector <LabelClass> initial_label_classes = cpu_gen_initial_labels(l0, l1, ring_classes);
    GpuLabelClass **gpu_initial_label_classes;
    int gpu_initial_label_classes_size = initial_label_classes.size();
    Pair m_local;
    vector <LabelClass> lcs;
    //cuda Mallocs
    //cuda malloc edge labels
    hipMallocManaged(&gpu_edge_labels, sizeof(float) * edge_labels.size());
    //cuda malloc adj matrix mol 0
    hipMallocManaged((void **) &gpu_g0, l0.size() * sizeof(float *));
    for (int i = 0; i < l0.size(); ++i) { hipMallocManaged((void **) &(gpu_g0[i]), l0.size() * sizeof(float)); }
    //cuda malloc adj matrix mol 1
    hipMallocManaged((void **) &gpu_g1, l1.size() * sizeof(float *));
    for (int i = 0; i < l1.size(); ++i) { hipMallocManaged((void **) &(gpu_g1[i]), l1.size() * sizeof(float)); }
    //cuda malloc GpuLabelClass array
    int size = initial_label_classes.size() * 2;
    hipMallocManaged(&gpu_initial_label_classes, initial_label_classes.size() * sizeof(GpuLabelClass *));
    for (int k = 0; k < initial_label_classes.size(); ++k) {
        hipMallocManaged(&gpu_initial_label_classes[k], sizeof(GpuLabelClass) * size);
        for (int i = 0; i < size; i++) {
            hipMallocManaged(&gpu_initial_label_classes[k][i].col_ring_size, sizeof(int *) * min_mol_size);
            hipMallocManaged(&gpu_initial_label_classes[k][i].g, sizeof(int) * l0.size());
            hipMallocManaged(&gpu_initial_label_classes[k][i].h, sizeof(int) * l1.size());
            hipMallocManaged(&gpu_initial_label_classes[k][i].rings_g, sizeof(int *) * min_mol_size);
            for (int j = 0; j < l0.size(); ++j) {
                hipMallocManaged(&(gpu_initial_label_classes[k][i].rings_g[j]), l0.size() * sizeof(int));}}}
    //cudamalloc / initialize pool
    hipMallocManaged(&thread_pool_read, sizeof(ThreadVar) * N);
    for (int j = 0; j < N; ++j) {
        thread_pool_read->labels_size = 0;
        thread_pool_read->m_size = 0;
        hipMallocManaged(&thread_pool_read[j].labels, sizeof(gpu_initial_label_classes));
        hipMallocManaged(&thread_pool_read[j].m_local, sizeof(Pair) * min_mol_size);
    }
    hipMallocManaged(&thread_pool_write, sizeof(ThreadVar) * N);
    for (int j = 0; j < N; ++j) {
        thread_pool_write->labels_size = 0;
        thread_pool_write->m_size = 0;
        hipMallocManaged(&thread_pool_write[j].labels, sizeof(gpu_initial_label_classes));
        hipMallocManaged(&thread_pool_write[j].m_local, sizeof(Pair) * min_mol_size);
    }

    //initialize
    //init edge labels
    vectorToPointerEdge(gpu_edge_labels);
    size_edge_labels = edge_labels.size();
    //init adj matrix mol0
    vectorToPointerMatrix(g0, gpu_g0);
    size_gpu_g0_row = g0.size();
    size_gpu_g0_col = g0[0].size();
    //init adj matrix mol 1
    vectorToPointerMatrix(g1, gpu_g1);
    size_gpu_g1_row = g1.size();
    size_gpu_g1_col = g1[0].size();

    cout << "CPU: Initializing thread pool" << endl;

    //init n_thread at 1
    int v,w,n_threads=1;
    for( LabelClass lc : initial_label_classes ) {
        v = select_vertex(lc.g,g0);
        w = select_vertex(lc.h,g1);
        if( !matchable(v,w,lc ) ) continue;
        m_local.first = v;
        m_local.second = w;
        lcs = genNewLabels(v,w,initial_label_classes);
        LabelFromCpuToGpu(gpu_initial_label_classes[n_threads],lcs);
        thread_pool_read[n_threads].labels = gpu_initial_label_classes[n_threads];
        thread_pool_read[n_threads].labels_size = lcs.size();
        thread_pool_read[n_threads].m_size = 1;
        thread_pool_read[n_threads].m_local[0] = m_local;
        n_threads++;
    }

    LabelFromCpuToGpu(gpu_initial_label_classes[0],initial_label_classes);
    host_select_label(&thread_pool_write[0].single_label, gpu_initial_label_classes[0],0, initial_label_classes.size() );
    
    std::vector<LabelClass*> label_class_pointers;
    if( !initial_label_classes.empty() ) label_class_pointers.reserve(initial_label_classes.size()+1); // Reserve space for the pointers

    for (LabelClass& item : initial_label_classes) {
        label_class_pointers.push_back(&item); // Add the address of each element to the new vector
    }

    LabelClass* single_label_class_pointer;
    single_label_class_pointer = select_label_DC(label_class_pointers, 0);
    
    cout << "GPU: label selected : " << thread_pool_write[0].single_label.label << endl;
    cout << "CPU : label selected : " << single_label_class_pointer->label << endl;


    int v_tmp = select_vertex(single_label_class_pointer->g,g0);
    int w_tmp = select_vertex(single_label_class_pointer->h,g1);
    int *result_v;
    int *result_w;
    int *result_pos;
    hipMallocManaged(&result_v, sizeof(int));
    hipMallocManaged(&result_w, sizeof(int));
    hipMallocManaged(&result_pos, sizeof(int));
    host_select_vertex(result_v,result_pos, thread_pool_write[0].single_label.g,  thread_pool_write[0].single_label.g_size, gpu_g0, size_gpu_g0_row, size_gpu_g0_col );
    host_select_vertex(result_w,result_pos, thread_pool_write[0].single_label.h,  thread_pool_write[0].single_label.h_size, gpu_g1, size_gpu_g1_row, size_gpu_g1_col );
    

    cout << "GPU: vertex V selected : " << *result_v << endl;
    cout << "CPU: vertex V selected : " << v_tmp << endl;

    cout << "GPU: vertex W selected : " << *result_w << endl;
    cout << "CPU: vertex W selected : " << w_tmp << endl;


    lcs = genNewLabels(v_tmp,w_tmp,initial_label_classes);
    int *idxList;
    hipMallocManaged(&idxList, sizeof(int) * min_mol_size);
    int new_labels_size = host_gen_new_Labels(thread_pool_write[1].labels, v_tmp, w_tmp, gpu_initial_label_classes[0], initial_label_classes.size(), idxList);

    
    //stampa thread pool read
    /*for ( int j = 0 ; j < n_threads ; ++j ){
        for ( int k = 0 ; k < thread_pool_read[j].labels_size ; ++k ){
            cout<< "\nLABEL : "<<thread_pool_read[j].labels[k].label;
            cout<< "\nADJ : "<<thread_pool_read[j].labels[k].adj;
            cout<<"\n G :  ";
            for ( int row = 0 ; row < thread_pool_read[j].labels[k].g_size ; row++ ){
                cout<<"[ "<< thread_pool_read[j].labels[k].g[row]<<" ]";
            }
            cout<<"\n H :  ";
            for ( int row = 0 ; row < thread_pool_read[j].labels[k].h_size ; row++ ){
                cout<<"[ "<< thread_pool_read[j].labels[k].h[row]<<" ]";
            }
            cout<<"\n ALL RINGS :  ";
            for ( int row = 0 ; row < thread_pool_read[j].labels[k].row_ring_size ; row++ ){
                cout<<"\n{{ ";
                for ( int col = 0 ; col < thread_pool_read[j].labels[k].col_ring_size[row] ; col++ ){
                    cout<<"[ "<< thread_pool_read[j].labels[k].rings_g[row][col] <<" ]";
                }
                cout<<" }} ";
            }
        }
    }*/

    size_t threadsPerBlock;

    size_t numberOfBlocks;

    threadsPerBlock = 8;
    numberOfBlocks = 8;
    int h = 0;
/*

    do{
        printf("\nnew kernel call\n");
    kernel_function<<< threadsPerBlock , numberOfBlocks >>>( thread_pool_read , thread_pool_write);
    hipDeviceSynchronize();
    
    cout<<"\n\n\nSTAMPA DI THPOOL WRITE POST CHIAMATA A FUNZIONE (stampo lo zero )\n";
    for( int i = 0; i < N ; i++){
        cout<<"[ "<< i <<"] "<<thread_pool_write[i].m_local->first<<endl;
    }
    cpyThreadPool( thread_pool_read , thread_pool_write );

    cout<<"\n\n\nSTAMPA DI THPOOL READ POST COPY (non stampo lo zero)\n";
    for( int i = 0; i < N ; i++){
        if( thread_pool_read[i].m_local->first > 0 )
            cout<<"[ "<<i<<"] "<<thread_pool_read[i].m_local->first<<endl;
    }
    /*int j = 0;
    for( int i = 0; i < N ; i++){ 
        if ( thread_pool_write[i].m_local->first != 0 ){
            thread_pool_read[j].m_local->first = thread_pool_write[i].m_local->first;
            j++;
        }
    }
   


    h++;
    }while( h < 2);


*/











    //hipFree
    hipFree( gpu_edge_labels );
    for (int i = 0; i < l0.size(); ++i) {hipFree(gpu_g0[i]);}
    hipFree(gpu_g0);
    for (int i = 0; i < l1.size(); ++i) {hipFree(gpu_g1[i]);}
    hipFree(gpu_g1);
    for ( int k = 0; k < initial_label_classes.size() ; ++k ) {
        for (int i = 0; i < size; i++) {
            hipFree(gpu_initial_label_classes[k][i].col_ring_size);
            hipFree(gpu_initial_label_classes[k][i].g);
            hipFree(gpu_initial_label_classes[k][i].h);
            for (int j = 0; j < l0.size(); ++j) {
                hipFree(gpu_initial_label_classes[k][i].rings_g[j]);
            }
            hipFree(gpu_initial_label_classes[k][i].rings_g);
        }
        hipFree(gpu_initial_label_classes[k]);
    }
    hipFree( gpu_initial_label_classes);

    vector<pair<int,int>> m;

    return m;
}