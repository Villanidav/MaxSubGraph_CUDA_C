#include "hip/hip_runtime.h"
//
// Created by davide on 5/3/24.
//
#include "test.hpp"
#include <vector>
#include <string.h>
#include <string>
#include <queue>
#include <algorithm>
#include <unordered_set>
using namespace std;



//struct 
typedef struct{
    int g_size;
    int h_size;
    int row_ring_size;
    int *col_ring_size;
    int *g;
    int *h;
    int adj;
    char label[4];
    int **rings_g;
}GpuLabelClass;


typedef struct{
    int first;
    int second;
}Pair;


typedef struct {
    int labels_size;
    int m_size;
    GpuLabelClass *labels;
    GpuLabelClass single_label;
    int *idxList;
    Pair *m_local;
}ThreadVar;


//global variables
const int DIM_POOL = 50;
std::vector<std::vector<float>> g00;
std::vector<std::vector<float>> g11;
std::vector<float> edge_labels;

__shared__ float *gpu_edge_labels;
__shared__ int size_edge_labels;


__shared__ float **gpu_g0;
__shared__ int size_gpu_g0_row;
__shared__ int size_gpu_g0_col;

__shared__ float **gpu_g1;
__shared__ int size_gpu_g1_row;
__shared__ int size_gpu_g1_col;


__shared__ Pair *m_best;
__shared__ int m_best_size;



//util functions
void checkError(hipError_t r) {
  if (r != hipSuccess) {
    printf("CUDA error on line %d: %s\n", hipGetErrorString(r));
    exit(0);
  }
}
int select_vertex(std::vector<int>& vtx_set, std::vector<std::vector<float> >& g) {
    // selects node from graph given a label, choosing an adjacent node with the maximum degree

    int max_deg = -1;
    int vtx = 0;
    for (int c_vtx : vtx_set) {
        int deg = 0;
        for (float i : g[c_vtx]) {
            if (i != 0) {
                deg += 1;
            }
        }

        if (deg > max_deg) {
            max_deg = deg;
            vtx = c_vtx;
        }
    }
    return vtx;
}
std::vector<std::string> find_common_strings(const std::vector<std::string>& l0, const std::vector<std::string>& l1) {
    // Utilizzare un set per memorizzare ed effettuare velocemente la ricerca di stringhe comuni uniche
    std::unordered_set<std::string> common_strings(l0.begin(), l0.end());

    // Vettore per memorizzare le stringhe comuni trovate
    std::vector<std::string> result;

    // Trovare le intersezioni tra le stringhe della seconda lista e le stringhe nel set
    for (const std::string& str : l1) {
        // Se la stringa è presente nel set delle stringhe comuni
        if (common_strings.find(str) != common_strings.end()) {
            // Aggiungila al risultato
            result.push_back(str);
            // Rimuovi la stringa dal set per evitare duplicati
            common_strings.erase(str);
        }
    }
    return result;
}
std::vector<LabelClass> cpu_gen_initial_labels(const std::vector<std::string>& l0, const std::vector<std::string>& l1,     std::vector<std::vector<int> >& ring_classes){
    std::vector<LabelClass> label_classes;
    const std::vector<string> common_labels = find_common_strings(l0,l1);


    for (const std::string& label : common_labels) {
        // Filter atoms and ring data based on label
        std::vector<int> g_elems;
        std::vector<std::vector<int> > g_ring_classes;
        for (size_t i = 0; i < l0.size(); ++i) {
            if (l0[i] == label) {
                g_elems.push_back(i);
                if( !ring_classes.empty() ) g_ring_classes.push_back(ring_classes[i]); // Assuming ring_classes access by index

            }
        }

        std::vector<int> h_elems;
        for (size_t j = 0; j < l1.size(); ++j) {
            if (l1[j] == label) {
                h_elems.push_back(j);
            }
        }

        LabelClass label_tmp(g_elems,h_elems,g_ring_classes,0, label);
        label_classes.push_back(label_tmp);

    }
    return label_classes;
}
std::vector<float> cpu_gen_bond_labels(const std::vector<std::vector<float> >& g0, const std::vector<std::vector<float> >& g1) {

    std::vector<float> all_labels;
    all_labels.push_back(0.0);
    all_labels.push_back(1.0);
    all_labels.push_back(1.5);
    all_labels.push_back(2.0);
    all_labels.push_back(3.0);
    all_labels.push_back(4.0);
    all_labels.push_back(5.0);
    all_labels.push_back(6.0);
    // Vector to store potential bond labels (extracted from both matrices)
    std::vector<float> intersection;

    // Iterate over rows and columns of g0
    for (size_t i = 0; i < g0.size(); ++i) {
        for (size_t j = 0; j < g0[i].size(); ++j) {
            float current_label = g0[i][j];

            // Check if the label exists in each row of g1 (avoid nested loops)
            bool found_in_g1 = false;
            for (const std::vector<float>& row : g1) {
                if (std::find(row.begin(), row.end(), current_label) != row.end()) {
                    found_in_g1 = true;
                    break;
                }
            }

            // If found in g1 and not already in the intersection, add it
            if (found_in_g1 && (intersection.empty() || intersection.back() != current_label)) {
                intersection.push_back(current_label);
            }
        }
    }


    // Sort the intersection vector for desired output
    std::sort(intersection.begin(), intersection.end());
    // Use unique to remove consecutive duplicates (may leave gaps)
    intersection.erase(std::unique(intersection.begin(), intersection.end()), intersection.end());

    // Resize the vector to remove empty space from erasing (optional)
    intersection.resize(intersection.size());

    // Return the intersection vector containing common bond labels (sorted and unique)
    return intersection;
}
std::vector<int> hood(int vtx, const std::vector<std::vector<float>>& g, float edge) {
    // Return the neighbors of a specified node, with the specified bond type.
    std::vector<int> friends;
    for (std::size_t i = 0; i < g.size(); ++i) {
        if (g[i][vtx] == edge && static_cast<std::size_t>(vtx) != i) {
            friends.push_back(i);
        }
    }

    return friends;
}
void printLabelClass(GpuLabelClass lb) {
        cout<< lb.label << " [ ";
        cout<< " G("<< lb.g_size << "): ";
        if(!lb.g_size == 0) {for ( int i = 0 ; i < lb.g_size; i++ ) cout<<"["<<lb.g[i]<<"]";}
        cout<< " H("<< lb.h_size << "): ";
        if(!lb.h_size == 0 ) {for ( int i = 0; i <  lb.h_size; i++ ) cout<<"["<<lb.h[i]<<"]";}
        cout<< " RINGS("<< lb.row_ring_size << "): [";
        for( int i = 0; i< lb.row_ring_size; i++){cout<<"("<<lb.col_ring_size[i]<<")"<<"["; for( int j = 0; j <  lb.col_ring_size[i]; j++) cout<<lb.rings_g[i][j]<<", ";  cout<<" ]";}
        cout<<"]";
        cout<< " edge : " <<lb.adj<<" " ;
        cout<< lb.label << " ] "<<endl;
}   
void copyIntArray(int *a, int *b, int sizeb){
    for ( int i = 0 ; i < sizeb ; i++){
        a[i] = b[i];
    }
}
void copyIntMatrix(int **a, int **b, int rowsize, int *colsize )
{

    for( int i = 0 ; i < rowsize ; i++){
        for( int j = 0 ; j < colsize[i] ; j++){
            a[i][j] = b[i][j];
        }
    }

}
void cpyGpuLabelClass(GpuLabelClass *l1, GpuLabelClass l2){
    l1->adj = l2.adj;
    l1->row_ring_size = l2.row_ring_size;
    l1->g_size = l2.g_size;
    l1->h_size = l2.h_size;
    strcpy(l1->label , l2.label);
    copyIntArray( l1->g , l2.g, l2.g_size);
    copyIntArray( l1->h, l2.h, l2.h_size);
    copyIntArray( l1->col_ring_size, l2.col_ring_size , l2.row_ring_size);
    copyIntMatrix( l1->rings_g, l2.rings_g, l2.row_ring_size, l2.col_ring_size );/**/
}
void vectorToPointerEdge(float *gpu_edge_labels){
    if(edge_labels.size() == 0){
        gpu_edge_labels = nullptr;
        return;
    }
    int size = 0;
    for(float edg : edge_labels){
        gpu_edge_labels[size] = edg;
        size++;
    }

    return;
}
void vectorToPointerMatrix(const std::vector<std::vector<float>>& g,float** gpu_g) {
    // Get dimensions of the vector
    int numRows = g.size();
    if (numRows == 0) {
        // Empty vector, set pointers to nullptr
        gpu_g = nullptr;
        return;
    }

    int numCol = g[0].size();

    for (int i = 0; i < numRows; ++i) {
        for (int j = 0; j < numCol; ++j) {
            gpu_g[i][j] = g[i][j];
        }
    }
}
void LabelFromCpuToGpu(GpuLabelClass *new_label, const vector<LabelClass>& old_label ){

    for (int idx = 0 ; idx < old_label.size() ; ++idx ){
        new_label[idx].g_size = old_label.at(idx).g.size();
        new_label[idx].h_size = old_label.at(idx).h.size();
        new_label[idx].row_ring_size = old_label.at(idx).rings_g.size();
        new_label[idx].adj = old_label.at(idx).adj;
        strcpy(new_label[idx].label, old_label.at(idx).label.c_str() );

        for ( int j = 0 ; j < old_label.at(idx).g.size() ; ++j ){
            new_label[idx].g[j] = old_label.at(idx).g.at(j);
        }
        for ( int j = 0 ; j < old_label.at(idx).h.size() ; ++j ){
            new_label[idx].h[j] = old_label.at(idx).h.at(j);}

        for ( int row = 0 ; row < old_label.at(idx).rings_g.size() ; ++row ){
            new_label[idx].col_ring_size[row] = old_label.at(idx).rings_g.at(row).size();
            for ( int col = 0 ; col < old_label.at(idx).rings_g.at(row).size() ; ++col ){
                new_label[idx].rings_g[row][col] = old_label.at(idx).rings_g.at(row).at(col);
            }
        }

    }
}
int cpyThreadPool( ThreadVar *thread_pool_read, ThreadVar *thread_pool_write ){
    int r_idx = 0;
    for( int w_idx = 0; w_idx < DIM_POOL*DIM_POOL ; w_idx++){ 
        thread_pool_read[r_idx].labels_size = thread_pool_write[w_idx].labels_size;
        //da modificare l'if
        if ( thread_pool_write[w_idx].labels_size > 0 ){
            thread_pool_read[r_idx].single_label.g_size = 0;
            thread_pool_read[r_idx].single_label.h_size = 0;
            thread_pool_read[r_idx].single_label.row_ring_size = 0;
            thread_pool_read[r_idx].m_size = thread_pool_write[w_idx].m_size;
            thread_pool_read[r_idx].m_local->first = thread_pool_write[w_idx].m_local->first;
            thread_pool_read[r_idx].m_local->second = thread_pool_write[w_idx].m_local->second;
            for ( int l_idx = 0 ; l_idx < thread_pool_write[w_idx].labels_size ; l_idx++ ){
                thread_pool_read[r_idx].labels[l_idx].adj = thread_pool_write[w_idx].labels[l_idx].adj;
                thread_pool_read[r_idx].labels[l_idx].row_ring_size = thread_pool_write[w_idx].labels[l_idx].row_ring_size;
                thread_pool_read[r_idx].labels[l_idx].g_size = thread_pool_write[w_idx].labels[l_idx].g_size;
                thread_pool_read[r_idx].labels[l_idx].h_size = thread_pool_write[w_idx].labels[l_idx].h_size;
                copyIntArray( thread_pool_read[r_idx].labels[l_idx].g , thread_pool_write[w_idx].labels[l_idx].g , thread_pool_write[w_idx].labels[l_idx].g_size );
                copyIntArray( thread_pool_read[r_idx].labels[l_idx].h , thread_pool_write[w_idx].labels[l_idx].h , thread_pool_write[w_idx].labels[l_idx].h_size );
                copyIntArray( thread_pool_read[r_idx].labels[l_idx].col_ring_size , thread_pool_write[w_idx].labels[l_idx].col_ring_size , thread_pool_write[w_idx].labels[l_idx].row_ring_size );
                for( int c = 0 ; c < 4 ; c++){
                    thread_pool_read[r_idx].labels[l_idx].label[c] = thread_pool_write[w_idx].labels[l_idx].label[c];
                }
                copyIntMatrix( thread_pool_read[r_idx].labels[l_idx].rings_g , thread_pool_write[w_idx].labels[l_idx].rings_g, 
                                thread_pool_write[w_idx].labels[l_idx].row_ring_size, thread_pool_write[w_idx].labels[l_idx].col_ring_size );
            }
            
            for( int ms = 0 ; ms < thread_pool_write[w_idx].m_size ; ms ++){
                thread_pool_read[r_idx].m_local[ms].first = thread_pool_write[w_idx].m_local[ms].first;
                 thread_pool_read[r_idx].m_local[ms].second = thread_pool_write[w_idx].m_local[ms].second;
            }
            thread_pool_read[r_idx].m_size = thread_pool_write[w_idx].m_size;
            r_idx++;
        }
        thread_pool_write[w_idx].labels_size = 0;
    }
    return r_idx;
}



// vtx_set: selected label class
// g: selected graph
bool host_contains(int value, int *arr, int size) {
    for (int i = 0; i < size; ++i) {
        if (arr[i] == value) {
            return true;
        }
    }
    return false;
}

//      puts into a 2D array the data regarding indexes of rings related to the array of elements
//      2D-array that will contain the result that will be modified
//      1D-array containing idxList
//      1D-array of elements
//      int size of elements
int host_get_ring_match_data(int *dim_col, int **result, int *idxList, int *elems, int elem_size, GpuLabelClass *lc){
    int index;
    int idx_list_size= 0 ;


    for( int i = 0; i < elem_size ; ++i){
        index = 0;
        for( int j = 0 ; j < lc->g_size ; ++j ){
            if( lc->g[j] == elems[i] ) {idxList[idx_list_size] = index; idx_list_size++;}
            index++;
        }
    }


    for ( int i = 0  ; i < idx_list_size ; ++i  ){
        dim_col[i] = lc->col_ring_size[idxList[i]];
        for ( int j = 0 ; j < dim_col[i] ; ++j ){
            result[i][j] = lc->rings_g[idxList[i]][j];
        }
    }



    return idx_list_size;
}


// return the best select label given an array of labels
void host_select_label(GpuLabelClass *label , GpuLabelClass *lcs, int map_size, int lcs_size){
    int min = 999;
    int max ;
    for( int i = 0 ; i < lcs_size ; ++i ){
        //printf("LABEL CLASSES INTERNE[%d]\n", i);
        //printLabelClass(lcs[i]);
        if( lcs[i].adj == 1 || map_size == 0 ){
            if( lcs[i].g_size > lcs[i].h_size ) max = lcs[i].g_size;
            else max = lcs[i].h_size;
            //printf("\nMAX : %d\n", max);
            if( max < min ){
                
                min = max;
                cpyGpuLabelClass(label, lcs[i] );
            }
        }
    }
    return;
}

// compute the bound given a 1D array of struct GpuLabelClass and its size
int host_calc_bound(GpuLabelClass *lcs, int lc_size) {
    int bound = 0;
    for( int i = 0 ; i < lc_size ; ++i){
        if ( lcs[i].g_size > lcs[i].h_size ) bound = bound + lcs[i].h_size;
        else bound = bound + lcs[i].g_size;
    }
    return bound;
}


//return = size of the friends
//friend is the OUTPUT
int host_hoodG(int *friends,int vtx, float edge, float **g, int size_g) {
    int size = 0;
    
    for (int i = 0; i < size_g; i++) {
        if ( g[i][vtx] == edge && vtx != i) {
            friends[size] = i;
            size++;
        }
    }

    return size;
}


// result == size of generated label
// output is l_draft
// input : v
void host_resize(int *array, int size_arr, int place_availabel){
    int count = 0;
    /*bool flag = true;
    for(int i = 0; i < size_arr && flag ; ++i){
        if(array[i] == -1){continue;}
        if( i == count){ count ++; continue; }
        array[count] = array[i]; count++;
        if(count == place_availabel){flag = false;}
    }*/


    for( int i = 0 ; i < size_arr && place_availabel > 0 ; i++){
        if( array[i] != -1 ){
            place_availabel--;
            array[count] = array[i];
            count ++;
        }
    }
}


int host_gen_new_labels(GpuLabelClass *l_draft ,  int v, int w, GpuLabelClass *lcs, int lcs_size, int *idxList) {
    int vs,ws, draft_size = 0;
    int dim_row;
    int count = 0;
    for ( int i = 0 ; i < lcs_size ; ++i ){
       
        for ( int j = 0 ; j < size_edge_labels ; ++j ){
            int friendsize;
            friendsize = host_hoodG( l_draft[draft_size].g , v , gpu_edge_labels[j], gpu_g0 , size_gpu_g0_row);
        
            vs = 0;
            for ( int k = 0; k < friendsize ; ++k ){
                if( host_contains(l_draft[draft_size].g[k] , lcs[i].g, lcs[i].g_size) ){ vs++;  }
                else{ l_draft[draft_size].g[k] = -1;}
            }
        
            host_resize(l_draft[draft_size].g, friendsize, vs );

            dim_row = host_get_ring_match_data(l_draft[draft_size].col_ring_size, l_draft[draft_size].rings_g, idxList ,l_draft[draft_size].g, vs, &lcs[i] );
            

            friendsize = host_hoodG(l_draft[draft_size].h, w, gpu_edge_labels[j], gpu_g1, size_gpu_g1_row );
            //printf("\n esco da hood 2");
            ws = 0;
            for ( int k = 0 ; k < friendsize ; ++k ){
                
                if( host_contains(l_draft[draft_size].h[k], lcs[i].h, lcs[i].h_size) ){  ws++; }
                else {
                    l_draft[draft_size].h[k] = -1;
                }
            }
            host_resize(l_draft[draft_size].h, friendsize, ws );
    
            int adj;
            if ( ws > 0 && vs > 0 ){
                if( gpu_edge_labels[j] != 0.0 || lcs[i].adj == 1 ) {adj = 1;}
                else { adj = 0; }

                l_draft[draft_size].g_size = vs;
                l_draft[draft_size].h_size = ws;
                l_draft[draft_size].row_ring_size = dim_row;
                l_draft[draft_size].adj = adj;
                for( int c = 0 ; c < 4 ; c++){
                    l_draft[draft_size].label[c] = lcs[i].label[c];
                }
                draft_size++;
            }
        }
    }
    return draft_size;
}


//given two atoms from the same label, return true if they are matchable, false otherwise
// based on how their rings matches
bool host_matchable(int **v_ring_atoms, int v, int w, GpuLabelClass *lc, int *idxList) {

    host_get_ring_match_data(lc->col_ring_size, v_ring_atoms, idxList , &v, 1 ,lc);
    if( lc->col_ring_size[idxList[0]] > 0  ){
        for(int i = 0; i < lc->col_ring_size[idxList[0]]  ; i++){
            if( v_ring_atoms[0][i] == -1 )return false;
            if( v_ring_atoms[0][i] == w ) return true;
        }
        return false;
    }
    return true;
}


// vtx_set: selected label class
// g: selected graph
void host_select_vertex(int *result, int *result_pos, int *vtx_set, int vtx_size, float **g, int num_row, int num_column) {
    int max_deg = -1;
    int vtx = 0;

    for(int i = 0; i < vtx_size; i++){
        int deg = 0;
        for(int j = 0; j < num_column; j++){
            int consider = g[vtx_set[i]][j];
            if(consider != 0){
                deg++;
            }
        }

        if(deg>max_deg){
            max_deg = deg;
            *result = vtx_set[i];
            *result_pos = i;
        }
    }
    return ;
}



//function used in the main -- equivalent to the ones above but using LabelClasses objects, not struct
bool matchable(const int v,const int w, LabelClass lc ) {
    std::vector<int> vector;
    vector.push_back(v);
    std::vector<int>  v_ring_atoms = {};
    v_ring_atoms = lc.get_ring_match_data(vector).at(0);

    if( !v_ring_atoms.empty() ) {
        for(const int x : v_ring_atoms){
            if( x == -1 )return false;
            if( x == w ) return true;}
        return false;
    }
    return true;
}

vector<LabelClass> genNewLabels(int v, int w, const vector<LabelClass>& lcs) {
    vector<LabelClass> l_draft;
    for(LabelClass label : lcs){

        for(float edge_l : edge_labels){
            std::vector<int> v_conn;
            std::vector<int> w_conn;
            std::vector<std::vector<int> > v_c_rings;

            for(int vtx : hood(v,g00,edge_l)){
                if( std::find(label.g.begin(),label.g.end(),vtx) != label.g.end() ){
                    v_conn.push_back(vtx);
                }
            }
            v_c_rings = label.get_ring_match_data(v_conn);

            for(int vtx : hood(w,g11,edge_l)){
                if(std::find(label.h.begin(),label.h.end(),vtx) != label.h.end() ){
                    w_conn.push_back(vtx);
                }
            }

            int adj;
            if(!v_conn.empty() && !w_conn.empty()){
                if(edge_l != 0.0 || label.adj == 1){
                    adj = 1;
                }else{
                    adj = 0;
                }
                LabelClass tmp(v_conn,w_conn,v_c_rings,adj, label.label);
                l_draft.push_back(tmp);
            }
        }
    }
    return l_draft;
}




void host_parallel_solve_mcs( ThreadVar *thread_pool_read, ThreadVar *thread_pool_write, int n_threads ){
    
    int index ;
    int z = 0;
    int space = 4;
    int flag = 0;
    
    for( index = 0 ; index < n_threads ; index++ ){
        GpuLabelClass *label = &thread_pool_read[index].single_label;
        host_select_label(label, thread_pool_read[index].labels, 
                            thread_pool_read[index].m_size, thread_pool_read[index].labels_size);
        
        flag = 0;
        if ( (thread_pool_read[index].m_size + host_calc_bound(thread_pool_read[index].labels, thread_pool_read[index].labels_size  )  
            < m_best_size) || !label  ) { flag = 1; }
        if( flag == 0 ){
            int jump = 0;
            for( int v_idx = 0 ; v_idx < label->g_size ; ++v_idx){
                for( int w_idx = 0 ; w_idx < label->h_size ; ++w_idx){

                    if( !host_matchable(label->rings_g, label->g[v_idx], label->h[w_idx], label, thread_pool_read[index].idxList ) ) continue;
                    
                    for( z = 0 ; z < thread_pool_read[index].m_size ; z++ ){
                        thread_pool_write[4*index + jump].m_local[z].first = thread_pool_read[index].m_local[z].first;
                        thread_pool_write[4*index + jump].m_local[z].second = thread_pool_read[index].m_local[z].second;
                    }
                    thread_pool_write[4*index + jump].m_size = thread_pool_read[index].m_size + 1;
                    thread_pool_write[4*index + jump].m_local[z].first = label->g[v_idx];
                    thread_pool_write[4*index + jump].m_local[z].second = label->h[v_idx];
                    

                    int l_s = host_gen_new_labels( thread_pool_write[(4*index + jump)].labels , label->g[v_idx], label->h[w_idx] ,
                                                                                            thread_pool_read[index].labels , thread_pool_read[index].labels_size ,
                                                                                            thread_pool_read[index].idxList);
                    
                    thread_pool_write[(4*index + jump)].labels_size = l_s;
                    if( thread_pool_write[4*index + jump].m_size > m_best_size ){
                        m_best_size = thread_pool_write[4*index + jump].m_size ;
                        for( int z = 0 ; z < thread_pool_write[4*index + jump].m_size ; z++ ){
                            m_best[z].first = thread_pool_write[4*index + jump].m_local[z].first;
                            m_best[z].second = thread_pool_write[4*index + jump].m_local[z].second;
                        }

                    }
                jump ++;
                }
            }
       }
    }
}



vector<pair<int,int>> gpu_mc_split(const std::vector<std::vector<float>>& g000, const std::vector<std::vector<float>>& g111,
                                   const std::vector<std::string>& l0, const std::vector<std::string>& l1,
                                   std::vector<std::vector<int> >& ring_classes) {
    g00 = g000;
    g11 = g111;
    size_t N = DIM_POOL * DIM_POOL;
    edge_labels = cpu_gen_bond_labels(g00, g11);
    std::vector <LabelClass> initial_label_classes = cpu_gen_initial_labels(l0, l1, ring_classes);

    
 
    //cuda Mallocs
    hipMallocManaged(&m_best , sizeof(Pair)* l1.size());
    //cuda malloc edge labels
    checkError(hipMallocManaged(&gpu_edge_labels, sizeof(float) * edge_labels.size()));
    //cuda malloc adj matrix mol 0
    checkError(hipMallocManaged((void **) &gpu_g0, l0.size() * sizeof(float *)));
    for (int i = 0; i < l0.size(); ++i) { checkError(hipMallocManaged((void **) &(gpu_g0[i]), l0.size() * sizeof(float))); }
    //cuda malloc adj matrix mol 1
    checkError(hipMallocManaged((void **) &gpu_g1, l1.size() * sizeof(float *)));
    for (int i = 0; i < l1.size(); ++i) { checkError(hipMallocManaged((void **) &(gpu_g1[i]), l1.size() * sizeof(float))); }
    //cuda malloc GpuLabelClass array
    int size = initial_label_classes.size() * 3;
    printf("INITIAL SIZE %d", size);
    int min_mol_size = std::min(l0.size(), l1.size());

    //cudamalloc / initialize pool
    ThreadVar *thread_pool_read;
    ThreadVar *thread_pool_write;

    checkError(hipMallocManaged(&thread_pool_read, sizeof(ThreadVar) * N/5));
    for (int j = 0; j < N/5; ++j) {

        checkError(hipMallocManaged(&thread_pool_read[j].single_label.col_ring_size, sizeof(int ) * min_mol_size));
        checkError(hipMallocManaged(&thread_pool_read[j].single_label.g, sizeof(int) * l0.size()));
        checkError(hipMallocManaged(&thread_pool_read[j].single_label.h, sizeof(int) * l1.size()));
        checkError(hipMallocManaged(&thread_pool_read[j].single_label.rings_g, sizeof(int *) * l0.size()));
        for (int h = 0; h < l0.size(); ++h) {
            checkError(hipMallocManaged(&(thread_pool_read[j].single_label.rings_g[h]), l0.size() * sizeof(int)));
            }
        checkError(hipMallocManaged(&thread_pool_read[j].m_local, sizeof(Pair) * min_mol_size));
        checkError(hipMallocManaged(&thread_pool_read[j].idxList, sizeof(int) * min_mol_size));

        checkError(hipMallocManaged(&thread_pool_read[j].labels, size * sizeof(GpuLabelClass)));
        for (int k = 0; k < size; k++) {
                checkError(hipMallocManaged(&thread_pool_read[j].labels[k].col_ring_size, sizeof(int ) * min_mol_size));
                checkError(hipMallocManaged(&thread_pool_read[j].labels[k].g, sizeof(int) * l0.size()));
                checkError(hipMallocManaged(&thread_pool_read[j].labels[k].h, sizeof(int) * l1.size()));
                checkError(hipMallocManaged(&thread_pool_read[j].labels[k].rings_g, sizeof(int *) * l0.size()));
                for (int h = 0; h < l0.size(); ++h) {
                    checkError(hipMallocManaged(&(thread_pool_read[j].labels[k].rings_g[h]), l0.size() * sizeof(int)));
                }
        }
    
    }
    checkError(hipMallocManaged(&thread_pool_write, sizeof(ThreadVar) * N));
    for (int j = 0; j < N; ++j) {
        checkError(hipMallocManaged(&thread_pool_write[j].labels, size * sizeof(GpuLabelClass)));
        
        for (int k = 0; k < size; k++) {
                checkError(hipMallocManaged(&thread_pool_write[j].labels[k].col_ring_size, sizeof(int ) * min_mol_size));
                checkError(hipMallocManaged(&thread_pool_write[j].labels[k].g, sizeof(int) * l0.size()));
                checkError(hipMallocManaged(&thread_pool_write[j].labels[k].h, sizeof(int) * l1.size()));
                checkError(hipMallocManaged(&thread_pool_write[j].labels[k].rings_g, sizeof(int *) * l0.size()));
                for (int h = 0; h < l0.size(); ++h) {
                    checkError(hipMallocManaged(&(thread_pool_write[j].labels[k].rings_g[h]), l0.size() * sizeof(int)));
                }
        }
        checkError(hipMallocManaged(&thread_pool_write[j].single_label.col_ring_size, sizeof(int ) * min_mol_size));
        checkError(hipMallocManaged(&thread_pool_write[j].single_label.g, sizeof(int) * l0.size()));
        checkError(hipMallocManaged(&thread_pool_write[j].single_label.h, sizeof(int) * l1.size()));
        checkError(hipMallocManaged(&thread_pool_write[j].single_label.rings_g, sizeof(int *) * l0.size()));
        for (int h = 0; h < l0.size(); ++h) {
            checkError(hipMallocManaged(&(thread_pool_write[j].single_label.rings_g[h]), l0.size() * sizeof(int)));
            }
        checkError(hipMallocManaged(&thread_pool_write[j].m_local, sizeof(Pair) * min_mol_size));
        checkError(hipMallocManaged(&thread_pool_write[j].idxList, sizeof(int) * min_mol_size));
    
    }

    //initialize
    //init edge labels
    vectorToPointerEdge(gpu_edge_labels);
    size_edge_labels = edge_labels.size();
    //init adj matrix mol0
    vectorToPointerMatrix(g00, gpu_g0);
    size_gpu_g0_row = g00.size();
    size_gpu_g0_col = g00[0].size();
    //init adj matrix mol 1
    vectorToPointerMatrix(g11, gpu_g1);
    size_gpu_g1_row = g11.size();
    size_gpu_g1_col = g11[0].size();
    Pair m_local;
    vector <LabelClass> lcs;
    cout << "CPU: Initializing thread pool" << endl;

    //init n_thread at 1
    int size_lcs = 0;
    int v,w,n_threads=0;
    for( LabelClass lc : initial_label_classes ) {
        v = select_vertex(lc.g,g00);
        w = select_vertex(lc.h,g11);
        if( !matchable(v,w,lc ) ) continue;
        m_local.first = v;
        m_local.second = w;
        lcs = genNewLabels(v,w,initial_label_classes);
        size_lcs = lcs.size();
        LabelFromCpuToGpu(thread_pool_read[n_threads].labels,lcs);
        thread_pool_read[n_threads].labels_size = lcs.size();
        thread_pool_read[n_threads].m_size = 1;
        thread_pool_read[n_threads].m_local[0] = m_local;
        n_threads++;
    }

    int max = n_threads;
    int level = 0;
    do{
        host_parallel_solve_mcs(thread_pool_read, thread_pool_write , n_threads);
        n_threads = cpyThreadPool( thread_pool_read , thread_pool_write );
        level++;
    }while( n_threads > 0 );

    vector<pair<int,int>> m;
    pair<int, int> tmp;
    for( int best = 0 ; best < m_best_size ; best++ ){
        tmp.first = m_best[best].first;
        tmp.second = m_best[best].second;
        m.push_back(tmp);
    }

    //hipFree
    hipFree( gpu_edge_labels );
    for (int i = 0; i < l0.size(); ++i) {hipFree(gpu_g0[i]);}
    hipFree(gpu_g0);
    for (int i = 0; i < l1.size(); ++i) {hipFree(gpu_g1[i]);}
    hipFree(gpu_g1);
    
    for (int j = 0; j < N; ++j) {
        for (int k = 0; k < size; k++) {
                checkError(hipFree(thread_pool_write[j].labels[k].col_ring_size));
                checkError(hipFree(thread_pool_write[j].labels[k].g));
                checkError(hipFree(thread_pool_write[j].labels[k].h));
                for (int h = 0; h < l0.size(); ++h) {
                    checkError(hipFree((thread_pool_write[j].labels[k].rings_g[h])));
                }
                checkError(hipFree(thread_pool_write[j].labels[k].rings_g));
        }
        checkError(hipFree(thread_pool_write[j].labels));

        checkError(hipFree(thread_pool_write[j].single_label.col_ring_size));
        checkError(hipFree(thread_pool_write[j].single_label.g));
        checkError(hipFree(thread_pool_write[j].single_label.h));
        for (int h = 0; h < l0.size(); ++h) {
            checkError(hipFree((thread_pool_write[j].single_label.rings_g[h])));
            }
        checkError(hipFree(thread_pool_write[j].single_label.rings_g));
        checkError(hipFree(thread_pool_write[j].m_local));
        checkError(hipFree(thread_pool_write[j].idxList));
    }
    checkError(hipFree(thread_pool_write));

    for (int j = 0; j < N/5; ++j) {
        for (int k = 0; k < size; k++) {
                checkError(hipFree(thread_pool_read[j].labels[k].col_ring_size));
                checkError(hipFree(thread_pool_read[j].labels[k].g));
                checkError(hipFree(thread_pool_read[j].labels[k].h));
                for (int h = 0; h < l0.size(); ++h) {
                    checkError(hipFree((thread_pool_read[j].labels[k].rings_g[h])));
                }
                checkError(hipFree(thread_pool_read[j].labels[k].rings_g));
        }
        checkError(hipFree(thread_pool_read[j].labels));

        checkError(hipFree(thread_pool_read[j].single_label.col_ring_size));
        checkError(hipFree(thread_pool_read[j].single_label.g));
        checkError(hipFree(thread_pool_read[j].single_label.h));
        for (int h = 0; h < l0.size(); ++h) {
            checkError(hipFree((thread_pool_read[j].single_label.rings_g[h])));
            }
        checkError(hipFree(thread_pool_read[j].single_label.rings_g));
        checkError(hipFree(thread_pool_read[j].m_local));
        checkError(hipFree(thread_pool_read[j].idxList));
    }
    checkError(hipFree(thread_pool_read));

    return m;
}

