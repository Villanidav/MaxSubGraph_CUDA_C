#include "hip/hip_runtime.h"
//
// Created by davide on 4/19/24.
//

#include <fstream>
#include <iostream>
#include <vector>
#include <string>
#include "test.hpp"
#include <ctime>
using namespace std;
using namespace RDKit;

void printMol(ROMol result) {
    std::vector<std::string> result_string;

    for (const auto &atom : result.atoms()) {
        result_string.push_back(atom->getSymbol());
    }

    cout << "[";
    for ( int idx = 0; idx < result_string.size(); idx++ ){
        if(idx == result_string.size()-1 ){
            cout <<"'"<<result_string.at(idx)<<"']"<<endl;
        }
        else cout <<"'"<<result_string.at(idx)<<"', ";
    }
}

int main(){

    ROMol result;
    std::string smile0 = "CN([C@@H]1C[C@@H](C1)C(=O)N2CCC2)c3ccc(cc3C(=O)n4cc(C(=O)O)c5ccccc45)[N+](=O)[O-]";
    std::string smile1 = "O=C([C@@H]1C[C@@H](C1)Nc2ccccc2C(=O)n3ccc4ccccc34)N5CCC5";
    hipFree(0);
    clock_t start = clock();
    result = smiles_mcs(smile0, smile1, 1,1);
    clock_t end = clock();
    double elapsed_seconds = (double)(end - start) / CLOCKS_PER_SEC;
    cout<<"\nElapsed time: [ " << elapsed_seconds << " ]"<<endl;
    printMol(result);

    return 0;
}
